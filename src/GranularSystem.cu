#include "hip/hip_runtime.h"
#include "CUDAFunctions.cuh"
#include "DArray.hpp"
#include "GranularParticles.hpp"
#include "GranularSystem.hpp"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <vector_functions.h>

GranularSystem::GranularSystem(
    std::shared_ptr<GranularParticles> &granular_particles,
    std::shared_ptr<GranularParticles> &boundary_particles,
    const float3 space_size, const float cell_length, const float dt,
    const float3 g, int3 cell_size, const float density)
    : _particles(std::move(granular_particles)),
      _boundaries(std::move(boundary_particles)), _solver(_particles),
      _space_size(space_size), _dt(dt), _g(g), _cell_length(cell_length),
      _cell_start_particle(cell_size.x * cell_size.y * cell_size.z + 1),
      _cell_start_boundary(cell_size.x * cell_size.y * cell_size.z + 1),
      _cell_size(cell_size),
      _buffer_int(
          std::max(total_size(), cell_size.x * cell_size.y * cell_size.z + 1)),
      _density(density), _max_mass(4), _min_mass(1),
      _buffer_boundary(_particles->size()) {
  // initalize the boundary_particles
  neighbor_search(_boundaries, _cell_start_boundary);
  // Set the mass of all the particles to 1
  thrust::fill(thrust::device, _particles->get_mass_ptr(),
               _particles->get_mass_ptr() + _particles->size(), 1);
  neighbor_search(_particles, _cell_start_particle);

  step();
}

void GranularSystem::neighbor_search(
    const std::shared_ptr<GranularParticles> &particles,
    DArray<int> &cell_start) {

  int num = particles->size();
  std::cout << "Starting neighbor search for " << num << " particles"
            << std::endl;

  // Debug check array sizes
  std::cout << "Buffer int size: " << _buffer_int.length() << std::endl;
  std::cout << "Cell start size: " << cell_start.length() << std::endl;

  // Verify memory state
  size_t free_mem, total_mem;
  hipMemGetInfo(&free_mem, &total_mem);
  // std::cout << "GPU Memory - Free: " << free_mem / 1024 / 1024
  //           << "MB, Total: " << total_mem / 1024 / 1024 << "MB" << std::endl;

  // Debug sync point before kernel launch
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "Pre-kernel sync error: " << hipGetErrorString(err)
              << std::endl;
    throw std::runtime_error("CUDA sync error before mapParticles2Cells");
  }

  // map the particles to their cell idx
  mapParticles2Cells_CUDA<<<(num - 1) / block_size + 1, block_size>>>(
      particles->get_particle_2_cell(), particles->get_pos_ptr(), _cell_length,
      _cell_size, num);

  // Check for kernel errors
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Kernel launch error: " << hipGetErrorString(err)
              << std::endl;
    throw std::runtime_error("CUDA kernel error in mapParticles2Cells");
  }

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "Kernel sync error: " << hipGetErrorString(err) << std::endl;
    throw std::runtime_error("CUDA sync error after mapParticles2Cells");
  }

  // copy the cell indexes to _buffer_int with error checking
  try {
    CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                         sizeof(int) * num, hipMemcpyDeviceToDevice));
  } catch (const std::exception &e) {
    std::cerr << "Error copying to buffer_int: " << e.what() << std::endl;
    throw;
  }

  // sort the position with the cell indexes
  try {
    thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                        _buffer_int.addr() + num, particles->get_pos_ptr());
  } catch (const std::exception &e) {
    std::cerr << "Error in sort_by_key: " << e.what() << std::endl;
    throw;
  }

  // copy the new sorted indexes back to _buffer_int
  try {
    CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                         sizeof(int) * num, hipMemcpyDeviceToDevice));
  } catch (const std::exception &e) {
    std::cerr << "Error copying back to buffer_int: " << e.what() << std::endl;
    throw;
  }

  // sort velocity based on the keys
  try {
    thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                        _buffer_int.addr() + num, particles->get_vel_ptr());
  } catch (const std::exception &e) {
    std::cerr << "Error in velocity sort_by_key: " << e.what() << std::endl;
    throw;
  }

  // sort mass based on the keys
  try {
    thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                        _buffer_int.addr() + num, particles->get_mass_ptr());
  } catch (const std::exception &e) {
    std::cerr << "Error in mass sort_by_key: " << e.what() << std::endl;
    throw;
  }

  // sort solver buffers based on the keys
  try {

    _solver.resize(particles->size());

    thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                        _buffer_int.addr() + num,
                        _solver.get_buffer_merge_ptr());

    thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                        _buffer_int.addr() + num,
                        _solver.get_buffer_merge_count_ptr());

    thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                        _buffer_int.addr() + num,
                        _solver.get_buffer_remove_ptr());
  } catch (const std::exception &e) {
    std::cerr << "Error in mass sort_by_key: " << e.what() << std::endl;
    throw;
  }

  // fill cell_start with zeroes
  try {
    thrust::fill(
        thrust::device, cell_start.addr(),
        cell_start.addr() + _cell_size.x * _cell_size.y * _cell_size.z + 1, 0);
  } catch (const std::exception &e) {
    std::cerr << "Error filling cell_start: " << e.what() << std::endl;
    throw;
  }

  // add number of particles per cell index to cell_start
  countingInCell_CUDA<<<(num - 1) / block_size + 1, block_size>>>(
      cell_start.addr(), particles->get_particle_2_cell(), num);

  // Check for kernel errors
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "countingInCell kernel error: " << hipGetErrorString(err)
              << std::endl;
    throw std::runtime_error("CUDA kernel error in countingInCell");
  }

  // calculate the prefix sum of cell_start
  try {
    thrust::exclusive_scan(thrust::device, cell_start.addr(),
                           cell_start.addr() +
                               _cell_size.x * _cell_size.y * _cell_size.z + 1,
                           cell_start.addr());
  } catch (const std::exception &e) {
    std::cerr << "Error in exclusive_scan: " << e.what() << std::endl;
    throw;
  }

  // Final sync point
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "Final sync error: " << hipGetErrorString(err) << std::endl;
    throw std::runtime_error("CUDA sync error at end of neighbor_search");
  }
}

float GranularSystem::step() {
  // hipEvent_t start, stop;
  // CUDA_CALL(hipEventCreate(&start));
  // CUDA_CALL(hipEventCreate(&stop));
  // CUDA_CALL(hipEventRecord(start, 0));

  neighbor_search(_particles, _cell_start_particle);
  try {
    _solver.step(_particles, _boundaries, _cell_start_particle,
                 _cell_start_boundary, _space_size, _cell_size, _cell_length,
                 _dt, _g, _density);
    // hipDeviceSynchronize();
    CHECK_KERNEL();
    set_surface_particles(_particles, _cell_start_particle);

    hipDeviceSynchronize();

    _solver.adaptive_sampling(_particles, _cell_start_particle, _max_mass,
                              _cell_size, _space_size, _cell_length, _density);

  } catch (const char *s) {
    std::cout << s << "\n";
  } catch (...) {
    std::cout << "Unknown Exception at " << __FILE__ << ": line" << __LINE__
              << "\n";
  }

  // float milliseconds;
  // CUDA_CALL(hipEventRecord(stop, 0));
  // CUDA_CALL(hipEventSynchronize(stop));
  // CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
  // CUDA_CALL(hipEventDestroy(start));
  // CUDA_CALL(hipEventDestroy(stop));
  // return milliseconds;
  return 1;
}

void GranularSystem::compute_boundary_mass() {
  // computeBoundaryMass_CUDA<<<(_boundaries->size() - 1) / block_size + 1,
  //                            block_size>>>(
  //     _boundaries->getMassPtr(), _boundaries->getPosPtr(),
  //     _boundaries->size(), cell_startBoundary.addr(), _cell_size,
  //     _sphCellLength, _sphRhoBoundary, _sphSmoothingRadius);
}

// __device__ void boundary_kernel(float *sum_kernel, const int i,
//                                 const int cell_id, float3 *pos, int
//                                 *cell_start, const int3 cell_size, const
//                                 float density) {
//   if (cell_id == (cell_size.x * cell_size.y * cell_size.z))
//     return;
//   auto j = cell_start[cell_id];
//   const auto end = cell_start[cell_id + 1];
//   while (j < end) {
//     *sum_kernel += cubic_spline_kernel(length(pos[i] - pos[j]), radius);
//     ++j;
//   }
//   return;
// }
//
// __global__ void computeBoundaryMass_CUDA(float *mass, float3 *pos,
//                                          const int num, int *cell_start,
//                                          const int3 cell_size,
//                                          const float cell_length,
//                                          const float rhoB, const float
//                                          radius) {
//   const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
//   if (i >= num)
//     return;
//   const auto cell_pos = make_int3(pos[i] / cell_length);
// #pragma unroll
//   for (auto m = 0; m < 27; ++m) {
//     const auto cellID = particlePos2cellIdx(
//         cell_pos + make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
//         cell_size);
//     boundary_kernel(&mass[i], i, cellID, pos, cell_start, cell_size, radius);
//   }
//   mass[i] = rhoB / fmaxf(EPSILON, mass[i]);
//   return;
// }

__global__ void find_surface(int *buffer_boundary, float3 *pos_granular,
                             const int num, int *cell_start_granular,
                             const int3 cell_size, const float cell_length) {
  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

  if (i >= num)
    return;
  __syncthreads();

  float3 centroid = make_float3(0.0f, 0.0f, 0.0f);
  unsigned int n_neighbors = 0;

  for (auto m = 0; m < 27; __syncthreads(), ++m) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos_granular[i] / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);
    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;
    // calculate centroid of neighbors
    int j = cell_start_granular[cellID];
    while (j < cell_start_granular[cellID + 1]) {
      if (i == j) {
        j++;
        continue;
      }
      centroid += pos_granular[j];
      n_neighbors++;
      j++;
    }
  }

  centroid /= (float)n_neighbors;

  float dis =
      norm3df(pos_granular[i].x - centroid.x, pos_granular[i].y - centroid.y,
              pos_granular[i].z - centroid.z);

  if (dis > 0.01 || n_neighbors < 5) {
    buffer_boundary[i] = 1;
  } else {
    buffer_boundary[i] = 0;
  }
}

void GranularSystem::set_surface_particles(
    const std::shared_ptr<GranularParticles> &particles,
    DArray<int> &cell_start) {
  const int num = particles->size();
  find_surface<<<(num - 1) / block_size + 1, block_size>>>(
      _buffer_boundary.addr(), particles->get_pos_ptr(), num,
      _cell_start_particle.addr(), _cell_size, _cell_length);

  CUDA_CALL(hipMemcpy(particles->get_surface_ptr(), _buffer_boundary.addr(),
                       sizeof(int) * particles->size(),
                       hipMemcpyDeviceToDevice));
}
