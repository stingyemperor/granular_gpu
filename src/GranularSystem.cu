#include "hip/hip_runtime.h"
#include "CUDAFunctions.cuh"
#include "DArray.hpp"
#include "GranularParticles.hpp"
#include "GranularSystem.hpp"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <memory>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <vector_functions.h>

GranularSystem::GranularSystem(
    std::shared_ptr<GranularParticles> &granular_particles,
    std::shared_ptr<GranularParticles> &boundary_particles,
    const float3 space_size, const float cell_length, const float dt,
    const float3 g, int3 cell_size, const int density)
    : _particles(std::move(granular_particles)),
      _boundaries(std::move(boundary_particles)), _solver(_particles),
      _space_size(space_size), _dt(dt), _g(g), _cell_length(cell_length),
      _cell_start_particle(cell_size.x * cell_size.y * cell_size.z + 1),
      _cell_start_boundary(cell_size.x * cell_size.y * cell_size.z + 1),
      _cell_size(cell_size),
      _buffer_int(
          std::max(total_size(), cell_size.x * cell_size.y * cell_size.z + 1)),
      _density(density), _buffer_boundary(_particles->size()) {
  // initalize the boundary_particles
  neighbor_search(_boundaries, _cell_start_boundary);
  // Set the mass of all the particles to 1
  thrust::fill(thrust::device, _particles->get_mass_ptr(),
               _particles->get_mass_ptr() + _particles->size(), 1);
  neighbor_search(_particles, _cell_start_particle);

  step();
}

void GranularSystem::neighbor_search(
    const std::shared_ptr<GranularParticles> &particles,
    DArray<int> &cell_start) {
  int num = particles->size();

  // map the particles to their cell idx
  mapParticles2Cells_CUDA<<<(num - 1) / block_size + 1, block_size>>>(
      particles->get_particle_2_cell(), particles->get_pos_ptr(), _cell_length,
      _cell_size, num);
  // copy the cell indexes to _buffer_int
  CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                       sizeof(int) * num, hipMemcpyDeviceToDevice));
  // sort the position with the cell indexes
  thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                      _buffer_int.addr() + num, particles->get_pos_ptr());
  // copy the new sorted indexes back to _buffer_int
  CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                       sizeof(int) * num, hipMemcpyDeviceToDevice));
  // sort velocity based on the keys
  thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                      _buffer_int.addr() + num, particles->get_vel_ptr());

  // fill cell_start with zeroes
  thrust::fill(
      thrust::device, cell_start.addr(),
      cell_start.addr() + _cell_size.x * _cell_size.y * _cell_size.z + 1, 0);

  // add number of particles per cell index to cell_start
  countingInCell_CUDA<<<(num - 1) / block_size + 1, block_size>>>(
      cell_start.addr(), particles->get_particle_2_cell(), num);
  // calculate the prefix sum of cell_start to help with neighbor search
  thrust::exclusive_scan(thrust::device, cell_start.addr(),
                         cell_start.addr() +
                             _cell_size.x * _cell_size.y * _cell_size.z + 1,
                         cell_start.addr());
  return;
}

float GranularSystem::step() {
  // hipEvent_t start, stop;
  // CUDA_CALL(hipEventCreate(&start));
  // CUDA_CALL(hipEventCreate(&stop));
  // CUDA_CALL(hipEventRecord(start, 0));

  neighbor_search(_particles, _cell_start_particle);
  try {
    _solver.step(_particles, _boundaries, _cell_start_particle,
                 _cell_start_boundary, _space_size, _cell_size, _cell_length,
                 _dt, _g, _density);
    hipDeviceSynchronize();
    CHECK_KERNEL();
  } catch (const char *s) {
    std::cout << s << "\n";
  } catch (...) {
    std::cout << "Unknown Exception at " << __FILE__ << ": line" << __LINE__
              << "\n";
  }

  set_surface_particles(_particles, _cell_start_particle);

  // float milliseconds;
  // CUDA_CALL(hipEventRecord(stop, 0));
  // CUDA_CALL(hipEventSynchronize(stop));
  // CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
  // CUDA_CALL(hipEventDestroy(start));
  // CUDA_CALL(hipEventDestroy(stop));
  // return milliseconds;
  return 1;
}

void GranularSystem::compute_boundary_mass() {
  // computeBoundaryMass_CUDA<<<(_boundaries->size() - 1) / block_size + 1,
  //                            block_size>>>(
  //     _boundaries->getMassPtr(), _boundaries->getPosPtr(),
  //     _boundaries->size(), cell_startBoundary.addr(), _cell_size,
  //     _sphCellLength, _sphRhoBoundary, _sphSmoothingRadius);
}

// __device__ void boundary_kernel(float *sum_kernel, const int i,
//                                 const int cell_id, float3 *pos, int
//                                 *cell_start, const int3 cell_size, const
//                                 float density) {
//   if (cell_id == (cell_size.x * cell_size.y * cell_size.z))
//     return;
//   auto j = cell_start[cell_id];
//   const auto end = cell_start[cell_id + 1];
//   while (j < end) {
//     *sum_kernel += cubic_spline_kernel(length(pos[i] - pos[j]), radius);
//     ++j;
//   }
//   return;
// }
//
// __global__ void computeBoundaryMass_CUDA(float *mass, float3 *pos,
//                                          const int num, int *cell_start,
//                                          const int3 cell_size,
//                                          const float cell_length,
//                                          const float rhoB, const float
//                                          radius) {
//   const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
//   if (i >= num)
//     return;
//   const auto cell_pos = make_int3(pos[i] / cell_length);
// #pragma unroll
//   for (auto m = 0; m < 27; ++m) {
//     const auto cellID = particlePos2cellIdx(
//         cell_pos + make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
//         cell_size);
//     boundary_kernel(&mass[i], i, cellID, pos, cell_start, cell_size, radius);
//   }
//   mass[i] = rhoB / fmaxf(EPSILON, mass[i]);
//   return;
// }

__global__ void find_surface(int *buffer_boundary, float3 *pos_granular,
                             const int num, int *cell_start_granular,
                             const int3 cell_size, const float cell_length) {
  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

  if (i >= num)
    return;
  __syncthreads();

  float3 centroid = make_float3(0.0f, 0.0f, 0.0f);
  unsigned int n_neighbors = 0;

  for (auto m = 0; m < 27; __syncthreads(), ++m) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos_granular[i] / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);
    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;
    // calculate centroid of neighbors
    int j = cell_start_granular[cellID];
    while (j < cell_start_granular[cellID + 1]) {
      if (i == j) {
        j++;
        continue;
      }
      centroid += pos_granular[j];
      n_neighbors++;
      j++;
    }
  }

  centroid /= (float)n_neighbors;

  float dis =
      norm3df(pos_granular[i].x - centroid.x, pos_granular[i].y - centroid.y,
              pos_granular[i].z - centroid.z);

  if (dis > 0.01 || n_neighbors < 5) {
    buffer_boundary[i] = 1;
  } else {
    buffer_boundary[i] = 0;
  }
}

void GranularSystem::set_surface_particles(
    const std::shared_ptr<GranularParticles> &particles,
    DArray<int> &cell_start) {
  const int num = particles->size();
  find_surface<<<(num - 1) / block_size + 1, block_size>>>(
      _buffer_boundary.addr(), particles->get_pos_ptr(), num,
      _cell_start_particle.addr(), _cell_size, _cell_length);

  CUDA_CALL(hipMemcpy(particles->get_surface_ptr(), _buffer_boundary.addr(),
                       sizeof(int) * particles->size(),
                       hipMemcpyDeviceToDevice));
}
