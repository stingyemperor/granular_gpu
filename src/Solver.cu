#include "hip/hip_runtime.h"
#include "CUDAFunctions.cuh"
#include "Global.hpp"
#include "Solver.hpp"
#include "hip/hip_vector_types.h"
#include <algorithm>
#include <cstdio>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <hip/hip_vector_types.h>

__device__ __constant__ double pi = 3.14159265358979323846;

void print_darray_int(const DArray<int> &_num_constraints) {
  // Step 1: Allocate host memory
  const unsigned int length = _num_constraints.length();
  std::vector<int> host_array(length);

  // Step 2: Copy data from device to host
  CUDA_CALL(hipMemcpy(host_array.data(), _num_constraints.addr(),
                       sizeof(int) * length, hipMemcpyDeviceToHost));

  // Step 3: Print the data
  for (size_t i = 0; i < length; ++i) {
    std::cout << "Constraint[" << i << "] = " << host_array[i] << std::endl;
  }
}

void print_positions(const DArray<float3> &positions) {
  const unsigned int length = positions.length();
  std::vector<float3> host_array(length);

  CUDA_CALL(hipMemcpy(host_array.data(), positions.addr(),
                       sizeof(float3) * length, hipMemcpyDeviceToHost));

  for (size_t i = 0; i < length; ++i) {
    std::cout << "Position[" << i << "] = (" << host_array[i].x << ", "
              << host_array[i].y << ", " << host_array[i].z << ")" << std::endl;
  }
}

void Solver::step(std::shared_ptr<GranularParticles> &particles,
                  const std::shared_ptr<GranularParticles> &boundary,
                  const DArray<int> &cell_start_granular,
                  const DArray<int> &cell_start_boundary, float3 space_size,
                  int3 cell_size, float cell_length, float dt, float3 G,
                  const int density) {

  _buffer_int.resize(particles->size());
  // apply forces
  // update velocity
  add_external_force(particles, dt, G);
  update_particle_positions(particles, dt);

  // update_neighborhood(particles);
  // project constraints
  project(particles, boundary, cell_start_granular, cell_start_boundary,
          cell_size, space_size, cell_length, 5, density);

  // TODO: resize remaning stuff

  final_update(particles, dt);
}

// WARNING: Seems to cause issues with incorrect neighbors
void Solver::update_neighborhood(
    const std::shared_ptr<GranularParticles> &particles) {

  const int num = particles->size();
  CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                       sizeof(int) * num, hipMemcpyDeviceToDevice));
  // NOTE: might need to fix the value
  thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                      _buffer_int.addr() + num, particles->get_pos().addr());

  return;
}

void Solver::add_external_force(std::shared_ptr<GranularParticles> &particles,
                                float dt, float3 G) {
  const auto dv = dt * G;
  thrust::transform(thrust::device, particles->get_vel_ptr(),
                    particles->get_vel_ptr() + particles->size(),
                    particles->get_vel_ptr(), ThrustHelper::plus<float3>(dv));
}

struct predict_position_functor {
  float dt;

  predict_position_functor(float _dt) : dt(_dt) {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, float3> &t) const {
    const float3 &pos = thrust::get<0>(t);
    const float3 &vel = thrust::get<1>(t);
    return make_float3(pos.x + dt * vel.x, pos.y + dt * vel.y,
                       pos.z + dt * vel.z);
  }
};

void Solver::update_particle_positions(
    std::shared_ptr<GranularParticles> &particles, float dt) {
  // Assuming particles->get_pos_ptr() returns a pointer to the first element of
  // the position buffer and particles->get_vel_ptr() returns a pointer to the
  // first element of the velocity buffer

  // Create zip iterator for positions and velocities
  // We use a zip iterator because we need to loop through postions and
  // velocties together
  auto begin = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr(), particles->get_vel_ptr()));
  auto end = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr() + particles->size(),
                         particles->get_vel_ptr() + particles->size()));

  // Update positions by applying the 'update_position_functor' across the range
  thrust::transform(
      thrust::device, begin, end, _pos_t.addr(),
      // particles->get_pos_ptr(), // Output to the positions buffer
      predict_position_functor(dt));
}

struct final_velocity_functor {
  float dt;

  final_velocity_functor(float _dt) : dt(_dt) {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, float3> &t) const {
    const float dt_inv = 1 / dt;
    const float3 &pos = thrust::get<0>(t);
    const float3 &pos_t = thrust::get<1>(t);

    return make_float3(dt_inv * (pos_t.x - pos.x), dt_inv * (pos_t.y - pos.y),
                       dt_inv * (pos_t.z - pos.z));
  }
};

void Solver::final_update(std::shared_ptr<GranularParticles> &particles,
                          float dt) {

  // update velocity
  auto begin = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr(), _pos_t.addr()));
  auto end = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr() + particles->size(),
                         _pos_t.addr() + particles->size()));

  thrust::transform(
      thrust::device, begin, end, particles->get_vel_ptr(),
      // particles->get_pos_ptr(), // Output to the positions buffer
      final_velocity_functor(dt));

  // update position
  CUDA_CALL(hipMemcpy(particles->get_pos_ptr(), _pos_t.addr(),
                       sizeof(float3) * particles->size(),
                       hipMemcpyDeviceToDevice));
}

__device__ void boundary_constraint(float3 &del_p, int &n, const float3 pos_p,
                                    float3 *pos_b, int j, const int cell_end,
                                    const int density) {
  while (j < cell_end) {
    const float dis = norm3df(pos_p.x - pos_b[j].x, pos_p.y - pos_b[j].y,
                              pos_p.z - pos_b[j].z);

    const float mag = dis - 2.0 * 0.01;
    const float3 p_12 = pos_p - pos_b[j];
    if (mag < 0.0) {
      del_p -= (mag / dis) * p_12;
      n++;
    }
    ++j;
  }
  return;
}

__device__ void particles_constraint(float3 &del_p, int &n, int i,
                                     float3 *pos_p, float *m, int j,
                                     const int cell_end, const int density,
                                     int *n_constraints, float3 *delta_pos) {
  while (j < cell_end) {
    if (i != j) {
      const float3 p_12 = pos_p[i] - pos_p[j];
      const float inv_m_i = 1 / m[i];
      const float inv_m_j = 1 / m[j];
      const float inv_m_sum = 1.0 / (inv_m_i + inv_m_j);
      const float r_i = cbrtf((3 * m[i]) / (4 * pi * density));
      const float r_j = cbrtf((3 * m[j]) / (4 * pi * density));
      if (i == 0) {

        // printf("m_1 = %f, m_2 = %f\n", inv_m_i, inv_m_j);
        // printf("r_1 = %f, r_2 = %f\n", r_i, r_j);
        // printf("mass_sum = %f\n", inv_m_sum);
      }
      const float dis =
          norm3df(pos_p[i].x - pos_p[j].x, pos_p[i].y - pos_p[j].y,
                  pos_p[i].z - pos_p[j].z);
      const float mag = dis - (r_i + r_j);

      // TODO: add mass scaling
      if (mag < 0.0) {
        del_p -= inv_m_sum * inv_m_i * (mag / dis) * p_12;
        delta_pos[j] += inv_m_sum * inv_m_j * (mag / dis) * p_12;

        n++;
        n_constraints[j]++;
      }
    }
    ++j;
  }
  return;
}

__global__ void compute_delta_pos(float3 *delta_pos, int *n,
                                  float3 *pos_granular, float3 *pos_boundary,
                                  float *mass_granular, const int num,
                                  int *cell_start_granular,
                                  int *cell_start_boundary,
                                  const int3 cell_size, const float cell_length,
                                  const int density) {

  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

  // out of bounds
  if (i >= num)
    return;
  float3 dp = make_float3(0.0f);

  __syncthreads();

#pragma unroll
  // Loop through the 27 neighboring cells
  for (auto m = 0; m < 27; __syncthreads(), ++m) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos_granular[i] / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);
    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;
    // contributeDeltaPos_fluid(dp, i, pos_fluid, lambda, massFluid,
    //                          cellStartFluid[cellID], cellStartFluid[cellID +
    //                          1], radius);
    boundary_constraint(dp, n[i], pos_granular[i], pos_boundary,
                        cell_start_boundary[cellID],
                        cell_start_boundary[cellID + 1], density);

    particles_constraint(
        dp, n[i], i, pos_granular, mass_granular, cell_start_granular[cellID],
        cell_start_granular[cellID + 1], density, n, delta_pos);
  }

  delta_pos[i] = dp;
  return;
}

struct change_position_functor {

  change_position_functor() {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, int, float3> &t) const {
    const float3 &del_pos = thrust::get<0>(t);
    const int &n = max(1, thrust::get<1>(t));
    const float3 &pos_t = thrust::get<2>(t);
    return pos_t + del_pos / n;
  }
};

void Solver::project(std::shared_ptr<GranularParticles> &particles,
                     const std::shared_ptr<GranularParticles> &boundaries,
                     const DArray<int> &cell_start_granular,
                     const DArray<int> &cell_start_boundary, int3 cell_size,
                     float3 space_size, float cell_length, int max_iter,
                     const int density) {

  int iter = 0;
  const float3 zero = make_float3(0.0f, 0.0f, 0.0f);
  const int num = particles->size();
  while (iter < max_iter) {

    // reset delta p and num constraints
    thrust::device_ptr<float3> thrust_buffer_float_3 =
        thrust::device_pointer_cast(_buffer_float3.addr());
    thrust::fill(thrust_buffer_float_3, thrust_buffer_float_3 + num, zero);

    thrust::device_ptr<int> thrust_num_constraints =
        thrust::device_pointer_cast(_num_constraints.addr());

    thrust::fill(thrust_num_constraints, thrust_num_constraints + num, 0);

    // print_darray_int(_num_constraints);

    compute_delta_pos<<<(num - 1) / block_size + 1, block_size>>>(

        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr(),
        boundaries->get_pos_ptr(), particles->get_mass_ptr(), num,
        cell_start_granular.addr(), cell_start_boundary.addr(), cell_size,
        cell_length, density);
    //
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(
        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr()));

    auto end = thrust::make_zip_iterator(
        thrust::make_tuple(_buffer_float3.addr() + num,
                           _num_constraints.addr() + num, _pos_t.addr() + num));

    thrust::transform(
        thrust::device, begin, end, _pos_t.addr(),
        // particles->get_pos_ptr(), // Output to the positions buffer
        change_position_functor());

    iter++;
  }
}

__device__ void viable_merge(const int i, int j, const int cell_end,
                             const float *m, const int *surface,
                             const int *remove, const float max_mass, int &n,
                             int *n_indices) {
  while (j < cell_end) {
    if (i != j) {
      const bool mass_check = m[j] == max_mass;
      // Only add as viable neighbor if:
      // 1. Not a surface particle
      // 2. Not already marked for removal
      // 3. Not at max mass
      if (!(surface[j] == 1 || remove[j] == 1 || mass_check)) {
        if (n < 100) { // Bounds check
          n_indices[n] = j;
          atomicAdd(&n, 1);
        }
      }
    }
    ++j;
  }
}

__global__ void merge_gpu(const int num, float3 *pos_granular,
                          float *mass_granular, int *surface, int *remove,
                          int *cell_start_granular, float max_mass,
                          const int3 cell_size, const float cell_length) {
  // store the indices of viable neighbors
  int neighbor_indices[100]; // Consider making this dynamic if needed

  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

  // Bounds check
  if (i >= num)
    return;

  // if the particle is not a surface particle
  if (surface[i] == 0) {
    int n = 0;
    remove[i] = 1; // Mark for removal initially

    __syncthreads();

#pragma unroll
    // Loop through the 27 neighboring cells
    for (auto m = 0; m < 27; __syncthreads(), ++m) {
      const auto cellID = particlePos2cellIdx(
          make_int3(pos_granular[i] / cell_length) +
              make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
          cell_size);

      if (cellID == (cell_size.x * cell_size.y * cell_size.z))
        continue;

      // Add atomicMax to track array bounds
      int maxNeighbors = atomicMax(&n, n);
      if (maxNeighbors >= 100) {
        printf("Warning: Neighbor array overflow at particle %d\n", i);
        continue;
      }

      // get the number and indices of viable neighbors
      viable_merge(i, cell_start_granular[cellID],
                   cell_start_granular[cellID + 1], mass_granular, surface,
                   remove, max_mass, n, neighbor_indices);
    }

    // no viable neighbors
    if (n == 0) {
      remove[i] = 0; // Keep the particle
    } else {
      // Redistribute mass to neighbors
      float new_mass = mass_granular[i] / n;
      for (int m = 0; m < n && m < 100; m++) {
        int neighborIdx = neighbor_indices[m];
        if (neighborIdx >= 0 && neighborIdx < num) {
          atomicAdd(&mass_granular[neighborIdx], new_mass);
        }
      }
    }
  } else {
    remove[i] = 0; // Don't remove surface particles
  }
}

void Solver::adaptive_sampling(std::shared_ptr<GranularParticles> &particles,
                               const DArray<int> &cell_start_granular,
                               const float max_mass, int3 cell_size,
                               float3 space_size, float cell_length) {
  const int num = particles->size();

  // Zero out removal buffer before merge
  thrust::fill(thrust::device, _buffer_remove.addr(),
               _buffer_remove.addr() + num, 0);

  // Add debug prints
  std::vector<int> debug_buffer(num);
  hipMemcpy(debug_buffer.data(), _buffer_remove.addr(), sizeof(int) * num,
             hipMemcpyDeviceToHost);
  std::cout << "Pre-merge buffer state: ";
  for (int i = 0; i < std::min(10, num); i++) {
    std::cout << debug_buffer[i] << " ";
  }
  std::cout << std::endl;

  // Run merge kernel
  merge_gpu<<<(num - 1) / block_size + 1, block_size>>>(
      num, particles->get_pos_ptr(), particles->get_mass_ptr(),
      particles->get_surface_ptr(), _buffer_remove.addr(),
      cell_start_granular.addr(), max_mass, cell_size, cell_length);

  // Verify kernel execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Merge kernel error: " << hipGetErrorString(err) << std::endl;
    return;
  }

  // Debug print after merge
  hipMemcpy(debug_buffer.data(), _buffer_remove.addr(), sizeof(int) * num,
             hipMemcpyDeviceToHost);
  std::cout << "Post-merge buffer state: ";
  for (int i = 0; i < std::min(10, num); i++) {
    std::cout << debug_buffer[i] << " ";
  }
  std::cout << std::endl;

  // Debug print number of particles to remove
  int remove_count = std::count(debug_buffer.begin(), debug_buffer.end(), 1);
  std::cout << "Particles marked for removal: " << remove_count << " out of "
            << num << std::endl;

  // Synchronize before compact
  hipDeviceSynchronize();

  try {
    particles->remove_elements(_buffer_remove);
  } catch (const std::exception &e) {
    std::cerr << "Error during remove_elements: " << e.what() << std::endl;
  }
}
