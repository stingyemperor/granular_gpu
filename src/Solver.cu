#include "hip/hip_runtime.h"
#include "Global.hpp"
#include "Solver.hpp"
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include <hip/hip_vector_types.h>

void Solver::step(std::shared_ptr<GranularParticles> &particles,
                  const std::shared_ptr<GranularParticles> &boundary,
                  const DArray<int> &cell_start_fluid,
                  const DArray<int> &cell_start_boundary, float3 space_size,
                  int3 cell_size, float cell_length, float radius, float dt,
                  float3 G) {

  update_neighborhood(particles);

  // apply forces
  // update velocity
  add_external_force(particles, dt, G);
}

void Solver::update_neighborhood(
    const std::shared_ptr<GranularParticles> &particles) {

  const int num = particles->size();
  CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                       sizeof(int) * num, hipMemcpyDeviceToDevice));
  // NOTE: might need to fix the value
  thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                      _buffer_int.addr() + num, particles->get_pos().addr());

  return;
}

void Solver::add_external_force(std::shared_ptr<GranularParticles> &particles,
                                float dt, float3 G) {
  const auto dv = dt * G;
  thrust::transform(thrust::device, particles->get_vel_ptr(),
                    particles->get_vel_ptr() + particles->size(),
                    particles->get_vel_ptr(), ThrustHelper::plus<float3>(dv));
}
