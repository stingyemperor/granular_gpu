#include "hip/hip_runtime.h"
#include "CUDAFunctions.cuh"
#include "Global.hpp"
#include "Solver.hpp"
#include "hip/hip_vector_types.h"
#include <algorithm>
#include <cstdio>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <device_atomic_functions.h>
#include <stdatomic.h>
#include <thrust/count.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <hip/hip_vector_types.h>

__device__ __constant__ double pi = 3.14159265358979323846;
#define EPSILON_m 1e-4f // Small threshold for comparison

void print_darray_int(const DArray<int> &_num_constraints) {
  // Step 1: Allocate host memory
  const unsigned int length = _num_constraints.length();
  std::vector<int> host_array(length);

  // Step 2: Copy data from device to host
  CUDA_CALL(hipMemcpy(host_array.data(), _num_constraints.addr(),
                       sizeof(int) * length, hipMemcpyDeviceToHost));

  // Step 3: Print the data
  for (size_t i = 0; i < length; ++i) {
    std::cout << "Constraint[" << i << "] = " << host_array[i] << std::endl;
  }
}

void print_mass(const DArray<float> &mass) {
  // Step 1: Allocate host memory
  const unsigned int length = mass.length();
  std::vector<float> host_array(length);

  // Step 2: Copy data from device to host
  CUDA_CALL(hipMemcpy(host_array.data(), mass.addr(), sizeof(float) * length,
                       hipMemcpyDeviceToHost));

  // Step 3: Print the data
  for (size_t i = 0; i < length; ++i) {
    std::cout << "Mass[" << i << "] = " << host_array[i] << std::endl;
  }
}

void print_positions(const DArray<float3> &positions) {
  const unsigned int length = positions.length();
  std::vector<float3> host_array(length);

  CUDA_CALL(hipMemcpy(host_array.data(), positions.addr(),
                       sizeof(float3) * length, hipMemcpyDeviceToHost));

  for (size_t i = 0; i < length; ++i) {
    std::cout << "Position[" << i << "] = (" << host_array[i].x << ", "
              << host_array[i].y << ", " << host_array[i].z << ")" << std::endl;
  }
}

void Solver::step(std::shared_ptr<GranularParticles> &particles,
                  const std::shared_ptr<GranularParticles> &boundary,
                  const DArray<int> &cell_start_granular,
                  const DArray<int> &cell_start_boundary, float3 space_size,
                  int3 cell_size, float cell_length, float dt, float3 G,
                  const float density) {

  _buffer_int.resize(particles->size());
  // apply forces
  // update velocity
  add_external_force(particles, dt, G);
  update_particle_positions(particles, dt);

  // update_neighborhood(particles);
  // project constraints
  project(particles, boundary, cell_start_granular, cell_start_boundary,
          cell_size, space_size, cell_length, 5, density);

  // TODO: resize remaning stuff

  final_update(particles, dt);
}

// WARNING: Seems to cause issues with incorrect neighbors
void Solver::update_neighborhood(
    const std::shared_ptr<GranularParticles> &particles) {

  const int num = particles->size();
  CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                       sizeof(int) * num, hipMemcpyDeviceToDevice));
  // NOTE: might need to fix the value
  thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                      _buffer_int.addr() + num, particles->get_pos().addr());

  return;
}

void Solver::add_external_force(std::shared_ptr<GranularParticles> &particles,
                                float dt, float3 G) {
  const auto dv = dt * G;
  thrust::transform(thrust::device, particles->get_vel_ptr(),
                    particles->get_vel_ptr() + particles->size(),
                    particles->get_vel_ptr(), ThrustHelper::plus<float3>(dv));
}

struct predict_position_functor {
  float dt;

  predict_position_functor(float _dt) : dt(_dt) {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, float3> &t) const {
    const float3 &pos = thrust::get<0>(t);
    const float3 &vel = thrust::get<1>(t);
    return make_float3(pos.x + dt * vel.x, pos.y + dt * vel.y,
                       pos.z + dt * vel.z);
  }
};

void Solver::update_particle_positions(
    std::shared_ptr<GranularParticles> &particles, float dt) {
  // Assuming particles->get_pos_ptr() returns a pointer to the first element of
  // the position buffer and particles->get_vel_ptr() returns a pointer to the
  // first element of the velocity buffer

  // Create zip iterator for positions and velocities
  // We use a zip iterator because we need to loop through postions and
  // velocties together
  auto begin = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr(), particles->get_vel_ptr()));
  auto end = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr() + particles->size(),
                         particles->get_vel_ptr() + particles->size()));

  // Update positions by applying the 'update_position_functor' across the range
  thrust::transform(
      thrust::device, begin, end, _pos_t.addr(),
      // particles->get_pos_ptr(), // Output to the positions buffer
      predict_position_functor(dt));
}

struct final_velocity_functor {
  float dt;

  final_velocity_functor(float _dt) : dt(_dt) {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, float3> &t) const {
    const float dt_inv = 1 / dt;
    const float3 &pos = thrust::get<0>(t);
    const float3 &pos_t = thrust::get<1>(t);

    return make_float3(dt_inv * (pos_t.x - pos.x), dt_inv * (pos_t.y - pos.y),
                       dt_inv * (pos_t.z - pos.z));
  }
};

void Solver::final_update(std::shared_ptr<GranularParticles> &particles,
                          float dt) {

  // update velocity
  auto begin = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr(), _pos_t.addr()));
  auto end = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr() + particles->size(),
                         _pos_t.addr() + particles->size()));

  thrust::transform(
      thrust::device, begin, end, particles->get_vel_ptr(),
      // particles->get_pos_ptr(), // Output to the positions buffer
      final_velocity_functor(dt));

  // update position
  CUDA_CALL(hipMemcpy(particles->get_pos_ptr(), _pos_t.addr(),
                       sizeof(float3) * particles->size(),
                       hipMemcpyDeviceToDevice));
}

__device__ void boundary_constraint(float3 &del_p, int &n, int i,
                                    const float3 pos_p, float3 *pos_b, float *m,
                                    int j, const int cell_end,
                                    const int density) {
  while (j < cell_end) {
    const float dis = norm3df(pos_p.x - pos_b[j].x, pos_p.y - pos_b[j].y,
                              pos_p.z - pos_b[j].z);

    const float r_i = cbrtf((3 * m[i]) / (4 * pi * density));
    const float mag = dis - (0.01 + r_i);
    const float3 p_12 = pos_p - pos_b[j];
    if (mag < 0.0) {
      del_p -= (mag / dis) * p_12;
      n++;
    }
    ++j;
  }
  return;
}

__device__ void particles_constraint(float3 &del_p, int &n, int i,
                                     float3 *pos_p, float *m, int j,
                                     const int cell_end, const int density,
                                     int *n_constraints, float3 *delta_pos) {
  while (j < cell_end) {
    if (i != j) {
      const float3 p_12 = pos_p[i] - pos_p[j];
      const float inv_m_i = 1 / m[i];
      const float inv_m_j = 1 / m[j];
      const float inv_m_sum = 1.0 / (inv_m_i + inv_m_j);
      const float r_i = cbrtf((3 * m[i]) / (4 * pi * density));
      const float r_j = cbrtf((3 * m[j]) / (4 * pi * density));
      if (i == 0) {

        // printf("m_1 = %f, m_2 = %f\n", inv_m_i, inv_m_j);
        // printf("r_1 = %f, r_2 = %f\n", r_i, r_j);
        // printf("mass_sum = %f\n", inv_m_sum);
      }
      const float dis =
          norm3df(pos_p[i].x - pos_p[j].x, pos_p[i].y - pos_p[j].y,
                  pos_p[i].z - pos_p[j].z);
      const float mag = dis - (r_i + r_j);

      // TODO: add mass scaling
      if (mag < 0.0) {
        del_p -= inv_m_sum * inv_m_i * (mag / dis) * p_12;
        delta_pos[j] += inv_m_sum * inv_m_j * (mag / dis) * p_12;

        n++;
        n_constraints[j]++;
      }
    }
    ++j;
  }
  return;
}

__global__ void compute_delta_pos(float3 *delta_pos, int *n,
                                  float3 *pos_granular, float3 *pos_boundary,
                                  float *mass_granular, const int num,
                                  int *cell_start_granular,
                                  int *cell_start_boundary,
                                  const int3 cell_size, const float cell_length,
                                  const int density) {

  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

  // out of bounds
  if (i >= num)
    return;
  float3 dp = make_float3(0.0f);

  __syncthreads();

#pragma unroll
  // Loop through the 27 neighboring cells
  for (auto m = 0; m < 27; __syncthreads(), ++m) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos_granular[i] / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);
    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;
    // contributeDeltaPos_fluid(dp, i, pos_fluid, lambda, massFluid,
    //                          cellStartFluid[cellID], cellStartFluid[cellID +
    //                          1], radius);
    boundary_constraint(dp, n[i], i, pos_granular[i], pos_boundary,
                        mass_granular, cell_start_boundary[cellID],
                        cell_start_boundary[cellID + 1], density);

    particles_constraint(
        dp, n[i], i, pos_granular, mass_granular, cell_start_granular[cellID],
        cell_start_granular[cellID + 1], density, n, delta_pos);
  }

  delta_pos[i] = dp;
  return;
}

struct change_position_functor {

  change_position_functor() {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, int, float3> &t) const {
    const float3 &del_pos = thrust::get<0>(t);
    const int &n = max(1, thrust::get<1>(t));
    const float3 &pos_t = thrust::get<2>(t);
    return pos_t + del_pos / n;
  }
};

void Solver::project(std::shared_ptr<GranularParticles> &particles,
                     const std::shared_ptr<GranularParticles> &boundaries,
                     const DArray<int> &cell_start_granular,
                     const DArray<int> &cell_start_boundary, int3 cell_size,
                     float3 space_size, float cell_length, int max_iter,
                     const float density) {

  int iter = 0;
  int stab_iter = 0;
  const float3 zero = make_float3(0.0f, 0.0f, 0.0f);
  const int num = particles->size();

  while (stab_iter < 2) {

    // reset change in position and number of elements
    thrust::device_ptr<float3> thrust_buffer_float_3 =
        thrust::device_pointer_cast(_buffer_float3.addr());
    thrust::fill(thrust_buffer_float_3, thrust_buffer_float_3 + num, zero);

    thrust::device_ptr<int> thrust_num_constraints =
        thrust::device_pointer_cast(_num_constraints.addr());

    thrust::fill(thrust_num_constraints, thrust_num_constraints + num, 0);

    // calculate change in positon
    compute_delta_pos<<<(num - 1) / block_size + 1, block_size>>>(

        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr(),
        boundaries->get_pos_ptr(), particles->get_mass_ptr(), num,
        cell_start_granular.addr(), cell_start_boundary.addr(), cell_size,
        cell_length, density);

    // update the position
    auto begin_p = thrust::make_zip_iterator(
        thrust::make_tuple(_buffer_float3.addr(), _num_constraints.addr(),
                           particles->get_pos_ptr()));

    auto end_p = thrust::make_zip_iterator(thrust::make_tuple(
        _buffer_float3.addr() + num, _num_constraints.addr() + num,
        particles->get_pos_ptr() + num));

    thrust::transform(
        thrust::device, begin_p, end_p, particles->get_pos_ptr(),
        // particles->get_pos_ptr(), // Output to the positions buffer
        change_position_functor());

    // update chage in position
    auto begin_del_p = thrust::make_zip_iterator(thrust::make_tuple(
        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr()));

    auto end_del_p = thrust::make_zip_iterator(
        thrust::make_tuple(_buffer_float3.addr() + num,
                           _num_constraints.addr() + num, _pos_t.addr() + num));

    thrust::transform(
        thrust::device, begin_del_p, end_del_p, _pos_t.addr(),
        // particles->get_pos_ptr(), // Output to the positions buffer
        change_position_functor());

    stab_iter++;
  }

  while (iter < max_iter) {

    // reset delta p and num constraints
    thrust::device_ptr<float3> thrust_buffer_float_3 =
        thrust::device_pointer_cast(_buffer_float3.addr());
    thrust::fill(thrust_buffer_float_3, thrust_buffer_float_3 + num, zero);

    thrust::device_ptr<int> thrust_num_constraints =
        thrust::device_pointer_cast(_num_constraints.addr());

    thrust::fill(thrust_num_constraints, thrust_num_constraints + num, 0);

    // print_darray_int(_num_constraints);

    compute_delta_pos<<<(num - 1) / block_size + 1, block_size>>>(

        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr(),
        boundaries->get_pos_ptr(), particles->get_mass_ptr(), num,
        cell_start_granular.addr(), cell_start_boundary.addr(), cell_size,
        cell_length, density);
    //
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(
        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr()));

    auto end = thrust::make_zip_iterator(
        thrust::make_tuple(_buffer_float3.addr() + num,
                           _num_constraints.addr() + num, _pos_t.addr() + num));

    thrust::transform(
        thrust::device, begin, end, _pos_t.addr(),
        // particles->get_pos_ptr(), // Output to the positions buffer
        change_position_functor());

    iter++;
  }
}

__device__ void viable_merge(const int i, int j, const int cell_end,
                             const float *m, const int *surface,
                             const int *remove, const float max_mass, int &n,
                             int *n_indices) {
  while (j < cell_end) {
    if (i != j) {
      if (remove[j] == 1) {
        n++;
      }
    }
    ++j;
  }
}

// __global__ void merge_mark_gpu(const int num, float3 *pos_granular,
//                                float *mass_granular, int *surface, int
//                                *remove, float *merge, int
//                                *cell_start_granular, float max_mass, const
//                                int3 cell_size, const float cell_length) {
//
//   const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
//   // Bounds check
//   if (i >= num) {
//     return;
//   }
//
//   // Validate input data
//   if (!pos_granular || !mass_granular || !surface || !remove || !merge ||
//       !cell_start_granular) {
//     printf("Kernel error: null pointer detected\n");
//     return;
//   }
//
//   // Only merge if the particle is not a surface particle, not marked to be
//   // removed and not marked to be merged
//   if (surface[i] == 0 && atomicOr(&remove[i], 0) == 0) {
//     float closest_dis = 1000.0f;
//     int closest_index = -1;
//
// #pragma unroll
//     // Loop through the 27 neighboring cells
//     for (auto m = 0; m < 27; __syncthreads(), ++m) {
//       const auto cellID = particlePos2cellIdx(
//           make_int3(pos_granular[i] / cell_length) +
//               make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
//           cell_size);
//
//       if (cellID >= cell_size.x * cell_size.y * cell_size.z) {
//         printf("Invalid cell ID computed for particle %d\n", i);
//         return;
//       }
//
//       if (cellID == (cell_size.x * cell_size.y * cell_size.z))
//         continue;
//
//       int j = cell_start_granular[cellID];
//       while (j < cell_start_granular[cellID + 1] && j < num) {
//         // Add debug print when we encounter a merging particle
//         if (atomicOr(&remove[j], 0) != 0) {
//           printf("Particle %d encountered particle %d which has remove flag =
//           "
//                  "%d and mass = %f\n",
//                  i, j, remove[j], mass_granular[j]);
//         }
//         // if j is marked for removal or marked to be merged or a surface
//         // particle
//         if (j <= i || atomicOr(&remove[j], 0) != 0 || surface[j] == 1) {
//           j++;
//           continue;
//         }
//
//         const bool mass_check = mass_granular[j] <= max_mass -
//         mass_granular[i];
//
//         if (!mass_check) {
//           atomicAdd(&j, 1);
//           continue;
//         }
//
//         const float3 p_i = pos_granular[i];
//         const float3 p_j = pos_granular[j];
//
//         const float dis =
//             norm3df((p_i.x - p_j.x), (p_i.y - p_j.y), (p_i.z - p_j.z));
//
//         if (dis < closest_dis) {
//           closest_dis = dis;
//           atomicExch(&closest_index, j);
//         }
//         atomicAdd(&j, 1);
//       }
//     }
//
//     // we found a viable candidate
//     if (closest_index != -1) {
//       // Try to mark both particles atomically
//       if (atomicCAS(&remove[i], 0, -1) == 0) { // First try to mark i
//         if (atomicCAS(&remove[closest_index], 0, -1) ==
//             0) { // Then try to mark closest_index
//           // Both particles were successfully marked
//           atomicExch(&merge[closest_index], mass_granular[i]);
//           atomicExch(&merge[i], -mass_granular[i]);
//           printf("Setting up merge: particle %d (mass %.3f) -> particle %d "
//                  "(mass %.3f), delta: %f and %f\n",
//                  i, mass_granular[i], closest_index,
//                  mass_granular[closest_index], mass_granular[i],
//                  -mass_granular[i]);
//         } else {
//           // Failed to mark closest_index, revert i's marking
//           atomicExch(&remove[i], 0);
//         }
//       }
//     }
//
//     // Try to atomically acquire the merge lock
//     // if (atomicCAS(&remove[closest_index], 0, -1) == 0) {
//     //   atomicExch(&remove[i], 1);
//     //   // NOTE: ??
//     //   // atomicExch(&merge[i], closest_index);
//     //   merge[closest_index] = mass_granular[i];
//     // }
//   }
//   return;
// }

__global__ void merge_mark_gpu(const int num, float3 *pos_granular,
                               float *mass_granular, int *surface, int *remove,
                               float *merge, int *cell_start_granular,
                               float max_mass, const int3 cell_size,
                               const float cell_length) {
  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  if (i >= num)
    return;

  // Only non-surface particles that aren't already marked
  if (surface[i] != 0 || atomicOr(&remove[i], 0) != 0)
    return;

  float3 pos_i = pos_granular[i];
  float mass_i = mass_granular[i];
  float closest_dis = 1000.0f;
  int closest_index = -1;

#pragma unroll
  for (auto m = 0; m < 27; __syncthreads(), ++m) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos_i / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);

    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;

    int j = cell_start_granular[cellID];
    while (j < cell_start_granular[cellID + 1] && j < num) {
      if (j <= i || atomicOr(&remove[j], 0) != 0 || surface[j] != 0) {
        j++;
        continue;
      }

      // Only merge if combined mass is valid
      float combined_mass = mass_i + mass_granular[j];
      if (combined_mass > max_mass) {
        j++;
        continue;
      }

      float3 p_j = pos_granular[j];
      float dis = norm3df(pos_i.x - p_j.x, pos_i.y - p_j.y, pos_i.z - p_j.z);

      if (dis < closest_dis) {
        closest_dis = dis;
        closest_index = j;
      }
      j++;
    }
  }

  // Only proceed if we found a merge candidate
  if (closest_index != -1) {
    if (atomicCAS(&remove[i], 0, -1) == 0) {
      if (atomicCAS(&remove[closest_index], 0, -1) == 0) {
        atomicExch(&merge[closest_index], mass_i);
        atomicExch(&merge[i], -mass_i);
        printf("Merge set up: particle %d (mass %.3f) -> particle %d (mass "
               "%.3f)\n",
               i, mass_i, closest_index, mass_granular[closest_index]);
      } else {
        atomicExch(&remove[i], 0);
      }
    }
  }
}
__device__ bool isAlmostZero(float x) {
  return fabsf(x) < EPSILON_m; // fabsf for single precision
}

__global__ void merge_count_gpu(const int num, float *mass_del,
                                float *mass_granular, int *merge_count,
                                int *remove, const int blend_factor) {
  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  if (i >= num)
    return;

  if (remove[i] == -1) {
    int old_count = atomicAdd(&merge_count[i], 1);
    float delta = mass_del[i] / blend_factor;
    atomicAdd(&mass_granular[i], delta);

    // Debug
    printf("Particle %d: step %d/%d, mass %.3f, delta %.3f\n", i, old_count + 1,
           blend_factor, mass_granular[i], delta);

    if (old_count + 1 == blend_factor) {
      if (mass_del[i] < 0) {
        // Only mark for removal if this particle is giving mass
        atomicExch(&remove[i], 1);
        printf("Particle %d marked for removal (final mass %.3f)\n", i,
               mass_granular[i]);
      } else {
        atomicExch(&remove[i], 0);
        printf("Particle %d merge complete (final mass %.3f)\n", i,
               mass_granular[i]);
      }
      atomicExch(&merge_count[i], 0);
      atomicExch(&mass_del[i], 0.0f);
    }
  }
}

__global__ void split_gpu(const int num, float3 *pos_granular,
                          float *mass_granular, float3 *vel_granular,
                          int *surface, int *remove, float *merge,
                          int *cell_start_granular, float min_mass,
                          const int3 cell_size, float *split_mass,
                          float3 *split_pos, float3 *split_vel,
                          int &split_count, float density) {

  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  // Merging/removing particles cannot split
  if (i >= num || atomicOr(&remove[i], 0) != 0 || surface[i] != 1)
    return;

  if (mass_granular[i] >= 2 * min_mass) {

    const float new_mass = mass_granular[i] / 2;
    const float3 new_vel = vel_granular[i];

    // TODO: make the position selection random

    const float r_i = cbrtf((3 * mass_granular[i]) / (4 * pi * density));

    const float3 new_pos_1 = make_float3(pos_granular[i].x + r_i,
                                         pos_granular[i].y, pos_granular[i].z);
    const float3 new_pos_2 = make_float3(pos_granular[i].x - r_i,
                                         pos_granular[i].y, pos_granular[i].z);

    mass_granular[i] = new_mass;
    atomicExch(&split_mass[split_count], new_mass);

    vel_granular[i] = new_vel;
    split_vel[split_count] = new_vel;

    pos_granular[i] = new_pos_1;
    split_pos[split_count] = new_pos_2;

    atomicAdd(&split_count, 1);
    // split
  }
  return;
}

struct merge_functor {

  merge_functor() {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float, float> &t) const {
    const float &mass = thrust::get<0>(t);
  }
};

// Host function
void Solver::adaptive_sampling(std::shared_ptr<GranularParticles> &particles,
                               const DArray<int> &cell_start_granular,
                               const float max_mass, int3 cell_size,
                               float3 space_size, float cell_length,
                               const float density) {
  const int num = particles->size();
  if (num == 0)
    return;

  hipDeviceSynchronize();

  try {
    // old mass

    auto m_t = thrust::device_pointer_cast(particles->get_mass_ptr());

    const float old_mass =
        thrust::reduce(m_t, m_t + num, 0, thrust::plus<float>());

    // Launch kernel with error checking
    hipError_t err = hipSuccess;

    // Run merge kernel
    merge_mark_gpu<<<(num + block_size - 1) / block_size, block_size>>>(
        num, particles->get_pos_ptr(), particles->get_mass_ptr(),
        particles->get_surface_ptr(), _buffer_remove.addr(),
        _buffer_merge.addr(), cell_start_granular.addr(), max_mass, cell_size,
        cell_length);

    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "Kernel launch failed: " << hipGetErrorString(err)
                << std::endl;
      throw std::runtime_error("Kernel launch failed");
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      std::cerr << "Kernel execution failed: " << hipGetErrorString(err)
                << std::endl;
      throw std::runtime_error("Kernel execution failed");
    }

    // Print info about particles being removed
    std::vector<int> host_remove(num);
    std::vector<float> host_mass(num);
    CUDA_CALL(hipMemcpy(host_remove.data(), _buffer_remove.addr(),
                         sizeof(int) * num, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(host_mass.data(), particles->get_mass_ptr(),
                         sizeof(float) * num, hipMemcpyDeviceToHost));

    int removal_count = 0;
    std::cout << "Particles marked for removal (mass):\n";
    for (int i = 0; i < num; i++) {
      if (host_remove[i] == 1) {
        removal_count++;
        std::cout << "Particle " << i << ": mass = " << host_mass[i]
                  << ", remove flag = " << host_remove[i]
                  << ", merge mass delta = ";

        // Get merge mass delta for this particle
        float merge_delta;
        CUDA_CALL(hipMemcpy(&merge_delta, &_buffer_merge.addr()[i],
                             sizeof(float), hipMemcpyDeviceToHost));
        std::cout << merge_delta << "\n";
      }
    }
    std::cout << "Total particles marked for removal: " << removal_count
              << "\n";

    // Run split kernel
    int n_split = 0;
    DArray<float> split_mass(1000);
    DArray<float3> split_pos(1000);
    DArray<float3> split_vel(1000);

    split_gpu<<<(num + block_size - 1) / block_size, block_size>>>(
        num, particles->get_pos_ptr(), particles->get_mass_ptr(),
        particles->get_vel_ptr(), particles->get_surface_ptr(),
        _buffer_remove.addr(), _buffer_merge.addr(), cell_start_granular.addr(),
        max_mass, cell_size, split_mass.addr(), split_pos.addr(),
        split_vel.addr(), n_split, density);

    hipDeviceSynchronize();

    split_mass.resize(n_split);
    split_pos.resize(n_split);
    split_vel.resize(n_split);

    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
      throw std::runtime_error(std::string("Merge kernel error: ") +
                               hipGetErrorString(err));
    }

    // TODO: Velocity update
    // thrust::transform(thrust::device, particles->get_mass_ptr(),
    //                   particles->get_mass_ptr() + num,
    //                   _buffer_merge.addr(), particles->get_mass_ptr(),
    //                   thrust::plus<float>());
    //

    // gradual merging
    merge_count_gpu<<<(num + block_size - 1) / block_size, block_size>>>(
        num, _buffer_merge.addr(), particles->get_mass_ptr(),
        _buffer_merge_count.addr(), _buffer_remove.addr(), _blend_factor);

    CUDA_CALL(hipDeviceSynchronize());

    // Print final state before removal
    std::cout << "\nFinal state before removal:\n";
    CUDA_CALL(hipMemcpy(host_mass.data(), particles->get_mass_ptr(),
                         sizeof(float) * num, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(host_remove.data(), _buffer_remove.addr(),
                         sizeof(int) * num, hipMemcpyDeviceToHost));

    for (int i = 0; i < num; i++) {
      if (host_remove[i] == 1) {
        std::cout << "Particle " << i << ": final mass = " << host_mass[i]
                  << "\n";
      }
    }

    // remove elements
    particles->remove_elements(_buffer_remove);
    _buffer_merge_count.compact(_buffer_remove);
    _buffer_merge.compact(_buffer_remove);
    _buffer_remove.compact(_buffer_remove);

    CUDA_CALL(hipDeviceSynchronize());

    // add elements
    particles->add_elements(split_mass, split_pos, split_vel, n_split);
    _buffer_merge_count.resize(particles->size());
    _buffer_merge.resize(particles->size());
    _buffer_remove.resize(particles->size());

    hipDeviceSynchronize();

    // change in mass

    // Print total mass after
    auto m_t_n = thrust::device_pointer_cast(particles->get_mass_ptr());
    const float new_mass = thrust::reduce(m_t_n, m_t_n + particles->size(), 0,
                                          thrust::plus<float>());
    std::cout << "Total mass after: " << new_mass << "\n";
    if ((new_mass - old_mass) != 0) {
      std::cout << "Change in mass " << new_mass - old_mass << "\n";
    }

  } catch (const std::exception &e) {
    std::cerr << "Error in adaptive_sampling: " << e.what() << std::endl;
    return;
  }
}
