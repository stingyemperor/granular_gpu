#include "hip/hip_runtime.h"
#include "CUDAFunctions.cuh"
#include "DArray.hpp"
#include "GranularParticles.hpp"
#include "GranularSystem.hpp"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <memory>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/sort.h>

GranularSystem::GranularSystem(
    std::shared_ptr<GranularParticles> &granular_particles,
    std::shared_ptr<GranularParticles> &boundary_particles,
    const float3 space_size, const float cell_length, const float dt,
    int3 cell_size)
    : _particles(std::move(granular_particles)),
      _boundaries(std::move(boundary_particles)), _solver(_particles),
      _space_size(space_size), _dt(dt), _cell_length(cell_length),
      _cell_start_particle(cell_size.x * cell_size.y * cell_size.z + 1),
      _cell_start_boundary(cell_size.x * cell_size.y * cell_size.z + 1),
      _cell_size(cell_size),
      _buffer_int(
          std::max(total_size(), cell_size.x * cell_size.y * cell_size.z + 1)) {
  neighbor_search(_boundaries, _cell_start_boundary);
  compute_boundary_mass();

  // Set the mass of all the particles to 1
  thrust::fill(thrust::device, _particles->get_mass_ptr(),
               _particles->get_mass_ptr() + _particles->size(), 1);
  neighbor_search(_particles, _cell_start_particle);

  step();
}

void GranularSystem::neighbor_search(
    const std::shared_ptr<GranularParticles> &particles,
    DArray<int> &cellStart) {
  int num = particles->size();

  // map the particles to their cell idx
  mapParticles2Cells_CUDA<<<(num - 1) / block_size + 1, block_size>>>(
      particles->get_particle_2_cell(), particles->get_pos_ptr(), _cell_length,
      _cell_size, num);
  // copy the cell indexes to _buffer_int
  CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                       sizeof(int) * num, hipMemcpyDeviceToDevice));
  // sort the position with the cell indexes
  thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                      _buffer_int.addr() + num, particles->get_pos_ptr());
  // copy the new sorted indexes back to _buffer_int
  CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                       sizeof(int) * num, hipMemcpyDeviceToDevice));
  // sort velocity based on the keys
  thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                      _buffer_int.addr() + num, particles->get_vel_ptr());

  // fill cell_start with zeroes
  thrust::fill(
      thrust::device, cellStart.addr(),
      cellStart.addr() + _cell_size.x * _cell_size.y * _cell_size.z + 1, 0);

  // add number of particles per cell index to cell_start
  countingInCell_CUDA<<<(num - 1) / block_size + 1, block_size>>>(
      cellStart.addr(), particles->get_particle_2_cell(), num);
  // calculate the prefix sum of cell_start to help with neighbor search
  thrust::exclusive_scan(thrust::device, cellStart.addr(),
                         cellStart.addr() +
                             _cell_size.x * _cell_size.y * _cell_size.z + 1,
                         cellStart.addr());
  return;
}

float GranularSystem::step() {
  // hipEvent_t start, stop;
  // CUDA_CALL(hipEventCreate(&start));
  // CUDA_CALL(hipEventCreate(&stop));
  // CUDA_CALL(hipEventRecord(start, 0));

  neighbor_search(_particles, _cell_start_particle);
  // try {
  // 	_solver->step(_fluids, _boundaries, cellStartFluid, cellStartBoundary,
  // 		_spaceSize, _cellSize, _sphCellLength, _sphSmoothingRadius,
  // 		_dt, _sphRho0, _sphRhoBoundary, _sphStiff, _sphVisc, _sphG,
  // 		_sphSurfaceTensionIntensity, _sphAirPressure);
  // 	hipDeviceSynchronize(); CHECK_KERNEL();
  // }
  // catch (const char* s) {
  // 	std::cout << s << "\n";
  // }
  // catch (...) {
  // 	std::cout << "Unknown Exception at "<<__FILE__<<": line "<<__LINE__ <<
  // "\n";
  // }

  // float milliseconds;
  // CUDA_CALL(hipEventRecord(stop, 0));
  // CUDA_CALL(hipEventSynchronize(stop));
  // CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
  // CUDA_CALL(hipEventDestroy(start));
  // CUDA_CALL(hipEventDestroy(stop));
  // return milliseconds;
  return 1;
}

void GranularSystem::compute_boundary_mass() {
  // computeBoundaryMass_CUDA<<<(_boundaries->size() - 1) / block_size + 1,
  //                            block_size>>>(
  //     _boundaries->getMassPtr(), _boundaries->getPosPtr(),
  //     _boundaries->size(), cellStartBoundary.addr(), _cellSize,
  //     _sphCellLength, _sphRhoBoundary, _sphSmoothingRadius);
}
