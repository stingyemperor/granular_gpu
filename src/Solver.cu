#include "hip/hip_runtime.h"
#include "CUDAFunctions.cuh"
#include "Global.hpp"
#include "GranularParticles.hpp"
#include "Solver.hpp"
#include "hip/hip_vector_types.h"
#include <algorithm>
#include <cstdio>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <device_atomic_functions.h>
#include <stdatomic.h>
#include <thrust/count.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <unistd.h>
#include <hip/hip_vector_types.h>

__device__ __constant__ double pi = 3.14159265358979323846;
__device__ __constant__ float r_9 = 493.8271605;
__device__ __constant__ float r_9_b = 1111.1111111;
// __device__ __constant__ float r_9 = 277.7777778;

#define EPSILON_m 1e-4f // Small threshold for comparison

int t_merge_iter = 0;
int t_iter_iter = 0;

void print_darray_int(const DArray<int> &_num_constraints) {
  // Step 1: Allocate host memory
  const unsigned int length = _num_constraints.length();
  std::vector<int> host_array(length);

  // Step 2: Copy data from device to host
  CUDA_CALL(hipMemcpy(host_array.data(), _num_constraints.addr(),
                       sizeof(int) * length, hipMemcpyDeviceToHost));

  // Step 3: Print the data
  for (size_t i = 0; i < length; ++i) {
    std::cout << "Constraint[" << i << "] = " << host_array[i] << std::endl;
  }
}

void print_mass(const DArray<float> &mass) {
  // Step 1: Allocate host memory
  const unsigned int length = mass.length();
  std::vector<float> host_array(length);

  // Step 2: Copy data from device to host
  CUDA_CALL(hipMemcpy(host_array.data(), mass.addr(), sizeof(float) * length,
                       hipMemcpyDeviceToHost));

  // Step 3: Print the data
  for (size_t i = -1; i < length; ++i) {
    std::cout << "Mass[" << i << "] = " << host_array[i] << std::endl;
  }
}

void print_positions(const DArray<float3> &positions) {
  const unsigned int length = positions.length();
  std::vector<float3> host_array(length);

  CUDA_CALL(hipMemcpy(host_array.data(), positions.addr(),
                       sizeof(float3) * length, hipMemcpyDeviceToHost));

  for (size_t i = 0; i < length; ++i) {
    std::cout << "Position[" << i << "] = (" << host_array[i].x << ", "
              << host_array[i].y << ", " << host_array[i].z << ")" << std::endl;
  }
}

void Solver::step(std::shared_ptr<GranularParticles> &particles,
                  const std::shared_ptr<GranularParticles> &boundary,
                  const DArray<int> &cell_start_granular,
                  const DArray<int> &cell_start_boundary, float3 space_size,
                  int3 cell_size, float cell_length, float dt, float3 G,
                  const float density) {

  _buffer_int.resize(particles->size());
  // apply forces
  // update velocity
  add_external_force(particles, dt, G);
  update_particle_positions(particles, dt);
  apply_mass_scaling(particles);

  // update_neighborhood(particles);
  // project constraints
  project(particles, boundary, cell_start_granular, cell_start_boundary,
          cell_size, space_size, cell_length, 5, density);

  // TODO: resize remaning stuff

  final_update(particles, dt);

  thrust::fill(
      thrust::device,
      thrust::device_pointer_cast(particles->get_adaptive_last_step_ptr()),
      thrust::device_pointer_cast(particles->get_adaptive_last_step_ptr() +
                                  particles->size()),
      0);
}

// WARNING: Seems to cause issues with incorrect neighbors
void Solver::update_neighborhood(
    const std::shared_ptr<GranularParticles> &particles) {

  const int num = particles->size();
  CUDA_CALL(hipMemcpy(_buffer_int.addr(), particles->get_particle_2_cell(),
                       sizeof(int) * num, hipMemcpyDeviceToDevice));
  // NOTE: might need to fix the value
  thrust::sort_by_key(thrust::device, _buffer_int.addr(),
                      _buffer_int.addr() + num, particles->get_pos().addr());

  return;
}

void Solver::add_external_force(std::shared_ptr<GranularParticles> &particles,
                                float dt, float3 G) {
  const auto dv = dt * G;
  thrust::transform(thrust::device, particles->get_vel_ptr(),
                    particles->get_vel_ptr() + particles->size(),
                    particles->get_vel_ptr(), ThrustHelper::plus<float3>(dv));
}

struct predict_position_functor {
  float dt;

  predict_position_functor(float _dt) : dt(_dt) {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, float3> &t) const {
    const float3 &pos = thrust::get<0>(t);
    const float3 &vel = thrust::get<1>(t);
    return make_float3(pos.x + dt * vel.x, pos.y + dt * vel.y,
                       pos.z + dt * vel.z);
  }
};

void Solver::update_particle_positions(
    std::shared_ptr<GranularParticles> &particles, float dt) {
  auto begin = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr(), particles->get_vel_ptr()));
  auto end = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr() + particles->size(),
                         particles->get_vel_ptr() + particles->size()));

  // Update positions by applying the 'update_position_functor' across the range
  thrust::transform(
      thrust::device, begin, end, _pos_t.addr(),
      // particles->get_pos_ptr(), // Output to the positions buffer
      predict_position_functor(dt));
}

struct mass_scaling_functor {
  // float _min_mass;
  // float _max_mass;
  // float _max_height;
  // float _min_height;

  // mass_scaling_functor(float min_mass, float max_mass, float max_height,
  //                      float min_height)
  //     : _min_mass(min_mass), _max_mass(max_mass), _min_height(min_height),
  //       _max_height(max_height) {}
  mass_scaling_functor() {}
  __host__ __device__ float
  operator()(const thrust::tuple<float, float3> &t) const {
    const float &mass = thrust::get<0>(t);
    const float3 &pos = thrust::get<1>(t);

    return mass * exp(-pos.y);
  }
};

void Solver::apply_mass_scaling(std::shared_ptr<GranularParticles> &particles) {
  auto begin = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_mass_ptr(), particles->get_pos_ptr()));
  auto end = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_mass_ptr() + particles->size(),
                         particles->get_pos_ptr() + particles->size()));

  thrust::transform(thrust::device, begin, end,
                    particles->get_scaled_mass_ptr(), mass_scaling_functor());
}

struct final_velocity_functor {
  float dt;
  float min_speed;   // Speed threshold below which damping is minimal
  float max_speed;   // Speed threshold for maximum damping
  float min_damping; // Minimum damping factor (for low velocities)
  float max_damping; // Maximum damping factor (for high velocities)

  final_velocity_functor(float _dt)
      : dt(_dt),
        min_speed(1.0f),    // Adjust these thresholds based on your simulation
        max_speed(10.0f),   // Adjust these thresholds based on your simulation
        min_damping(0.99f), // Almost no damping for slow particles
        max_damping(0.7f)   // Stronger damping for fast particles
  {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, float3, int> &t) const {
    const float dt_inv = 1.0f / dt;
    const float3 &pos = thrust::get<0>(t);
    const float3 &pos_t = thrust::get<1>(t);
    const int &adaptive_last = thrust::get<2>(t);

    // Calculate raw velocity
    float3 vel =
        make_float3(dt_inv * (pos_t.x - pos.x), dt_inv * (pos_t.y - pos.y),
                    dt_inv * (pos_t.z - pos.z));

    // Calculate speed
    float speed = length(vel);

    if (adaptive_last == 1) {
      vel *= 0.5f;
    } else {
      if (speed > min_speed) {
        // Calculate damping factor based on speed
        float t =
            clamp((speed - min_speed) / (max_speed - min_speed), 0.0f, 1.0f);

        // Smooth interpolation between min and max damping
        float damping = min_damping + (max_damping - min_damping) * t;

        // Apply non-linear damping
        float damping_factor = damping + (1.0f - damping) * expf(-speed * 0.1f);

        // Apply damping
        vel *= damping_factor;
      }
    }

    return vel;
  }
};
void Solver::final_update(std::shared_ptr<GranularParticles> &particles,
                          float dt) {

  // update velocity
  auto begin = thrust::make_zip_iterator(
      thrust::make_tuple(particles->get_pos_ptr(), _pos_t.addr(),
                         particles->get_adaptive_last_step_ptr()));
  auto end = thrust::make_zip_iterator(thrust::make_tuple(
      particles->get_pos_ptr() + particles->size(),
      _pos_t.addr() + particles->size(),
      particles->get_adaptive_last_step_ptr() + particles->size()));

  thrust::transform(
      thrust::device, begin, end, particles->get_vel_ptr(),
      // particles->get_pos_ptr(), // Output to the positions buffer
      final_velocity_functor(dt));

  // update position
  CUDA_CALL(hipMemcpy(particles->get_pos_ptr(), _pos_t.addr(),
                       sizeof(float3) * particles->size(),
                       hipMemcpyDeviceToDevice));
}

__device__ void boundary_constraint(float3 &del_p, int &n, int i,
                                    const float3 pos_p, float3 *pos_b, float *m,
                                    int j, const int cell_end,
                                    const int density) {
  while (j < cell_end) {
    const float dis = norm3df(pos_p.x - pos_b[j].x, pos_p.y - pos_b[j].y,
                              pos_p.z - pos_b[j].z);

    const float r_i = max(cbrtf((3 * m[i]) / (4 * pi * density)), 0.01f);
    const float mag = dis - (0.01 + r_i);
    const float3 p_12 = pos_p - pos_b[j];

    if (mag < 0.0) {
      const float3 del_p_i = (mag / dis) * p_12;
      const float3 del_p_i_perp =
          del_p_i - dot(del_p_i, p_12) * p_12 / (dis * dis);
      const float del_p_i_norm = norm3df(del_p_i.x, del_p_i.y, del_p_i.z);

      const float del_p_i_perp_norm =
          norm3df(del_p_i_perp.x, del_p_i_perp.y, del_p_i_perp.z);

      float min_fric = min((0.01 + r_i) * 0.8 / del_p_i_perp_norm, 1.0f);

      if (del_p_i_perp_norm < (r_i + 0.01) * 0.8) {
        del_p -= del_p_i;
      } else {
        del_p -= del_p_i * min_fric;
      }
      n++;
    }
    ++j;
  }
  return;
}

__device__ void particles_constraint(float3 &del_p, int &n, int i,
                                     float3 *pos_p, float *m, int j,
                                     const int cell_end, const int density,
                                     int *n_constraints, float3 *delta_pos) {
  while (j < cell_end) {
    if (i != j) {
      const float3 p_12 = pos_p[i] - pos_p[j];
      const float inv_m_i = 1 / m[i];
      const float inv_m_j = 1 / m[j];
      const float inv_m_sum = 1.0 / (inv_m_i + inv_m_j);
      const float r_i = cbrtf((3 * m[i]) / (4 * pi * density));
      const float r_j = cbrtf((3 * m[j]) / (4 * pi * density));
      const float dis =
          norm3df(pos_p[i].x - pos_p[j].x, pos_p[i].y - pos_p[j].y,
                  pos_p[i].z - pos_p[j].z);
      const float mag = (r_i + r_j) - dis;

      // TODO: add mass scaling
      if (mag >= 0.0) {
        // del_p -= inv_m_sum * inv_m_i * (mag / dis) * p_12;
        // delta_pos[j] += inv_m_sum * inv_m_j * (mag / dis) * p_12;

        const float3 del_p_i = -inv_m_sum * inv_m_i * (mag / dis) * p_12;
        const float3 del_p_j = inv_m_sum * inv_m_j * (mag / dis) * p_12;

        const float3 del_p_ij = del_p_i - del_p_j;
        const float3 del_p_ij_perp =
            del_p_ij - dot(del_p_ij, p_12) * p_12 / (dis * dis);

        const float del_p_ij_perp_norm =
            norm3df(del_p_ij_perp.x, del_p_ij_perp.y, del_p_ij_perp.z);

        const float min_fric =
            min((r_i + r_j) * 0.8 / del_p_ij_perp_norm, 1.0f);

        if (del_p_ij_perp_norm < (r_i + r_j) * 0.8) {
          del_p -= inv_m_sum * inv_m_i * del_p_ij;
          delta_pos[j] += inv_m_sum * inv_m_i * del_p_ij;
        } else {
          del_p -= inv_m_sum * inv_m_i * del_p_ij * min_fric;
          delta_pos[j] += inv_m_sum * inv_m_i * del_p_ij * min_fric;
        }

        n++;
        n_constraints[j]++;
      }
    }
    ++j;
  }
  return;
}

__global__ void compute_delta_pos(float3 *delta_pos, int *n,
                                  float3 *pos_granular, float3 *pos_boundary,
                                  float *mass_granular, const int num,
                                  int *cell_start_granular,
                                  int *cell_start_boundary,
                                  const int3 cell_size, const float cell_length,
                                  const int density) {

  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

  // out of bounds
  if (i >= num)
    return;
  float3 dp = make_float3(0.0f);

  __syncthreads();

#pragma unroll
  // Loop through the 27 neighboring cells
  for (auto m = 0; m < 27; __syncthreads(), ++m) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos_granular[i] / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);
    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;
    // contributeDeltaPos_fluid(dp, i, pos_fluid, lambda, massFluid,
    //                          cellStartFluid[cellID], cellStartFluid[cellID +
    //                          1], radius);
    boundary_constraint(dp, n[i], i, pos_granular[i], pos_boundary,
                        mass_granular, cell_start_boundary[cellID],
                        cell_start_boundary[cellID + 1], density);

    particles_constraint(
        dp, n[i], i, pos_granular, mass_granular, cell_start_granular[cellID],
        cell_start_granular[cellID + 1], density, n, delta_pos);
  }

  delta_pos[i] = dp;
  return;
}

struct change_position_functor {

  change_position_functor() {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float3, int, float3> &t) const {
    const float3 &del_pos = thrust::get<0>(t);
    const int &n = max(1, thrust::get<1>(t));
    const float3 &pos_t = thrust::get<2>(t);
    return pos_t + del_pos / n;
  }
};

void Solver::project(std::shared_ptr<GranularParticles> &particles,
                     const std::shared_ptr<GranularParticles> &boundaries,
                     const DArray<int> &cell_start_granular,
                     const DArray<int> &cell_start_boundary, int3 cell_size,
                     float3 space_size, float cell_length, int max_iter,
                     const float density) {

  int iter = 0;
  int stab_iter = 0;
  const float3 zero = make_float3(0.0f, 0.0f, 0.0f);
  const int num = particles->size();

  while (stab_iter < 3) {

    // reset change in position and number of elements
    thrust::device_ptr<float3> thrust_buffer_float_3 =
        thrust::device_pointer_cast(_buffer_float3.addr());
    thrust::fill(thrust_buffer_float_3, thrust_buffer_float_3 + num, zero);

    thrust::device_ptr<int> thrust_num_constraints =
        thrust::device_pointer_cast(_num_constraints.addr());

    thrust::fill(thrust_num_constraints, thrust_num_constraints + num, 0);

    // calculate change in positon
    compute_delta_pos<<<(num - 1) / block_size + 1, block_size>>>(

        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr(),
        boundaries->get_pos_ptr(), particles->get_scaled_mass_ptr(), num,
        cell_start_granular.addr(), cell_start_boundary.addr(), cell_size,
        cell_length, density);

    // update the position
    auto begin_p = thrust::make_zip_iterator(
        thrust::make_tuple(_buffer_float3.addr(), _num_constraints.addr(),
                           particles->get_pos_ptr()));

    auto end_p = thrust::make_zip_iterator(thrust::make_tuple(
        _buffer_float3.addr() + num, _num_constraints.addr() + num,
        particles->get_pos_ptr() + num));

    thrust::transform(
        thrust::device, begin_p, end_p, particles->get_pos_ptr(),
        // particles->get_pos_ptr(), // Output to the positions buffer
        change_position_functor());

    // update chage in position
    auto begin_del_p = thrust::make_zip_iterator(thrust::make_tuple(
        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr()));

    auto end_del_p = thrust::make_zip_iterator(
        thrust::make_tuple(_buffer_float3.addr() + num,
                           _num_constraints.addr() + num, _pos_t.addr() + num));

    thrust::transform(
        thrust::device, begin_del_p, end_del_p, _pos_t.addr(),
        // particles->get_pos_ptr(), // Output to the positions buffer
        change_position_functor());

    stab_iter++;
  }

  while (iter < max_iter) {

    // reset delta p and num constraints
    thrust::device_ptr<float3> thrust_buffer_float_3 =
        thrust::device_pointer_cast(_buffer_float3.addr());
    thrust::fill(thrust_buffer_float_3, thrust_buffer_float_3 + num, zero);

    thrust::device_ptr<int> thrust_num_constraints =
        thrust::device_pointer_cast(_num_constraints.addr());

    thrust::fill(thrust_num_constraints, thrust_num_constraints + num, 0);

    // print_darray_int(_num_constraints);

    compute_delta_pos<<<(num - 1) / block_size + 1, block_size>>>(

        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr(),
        boundaries->get_pos_ptr(), particles->get_mass_ptr(), num,
        cell_start_granular.addr(), cell_start_boundary.addr(), cell_size,
        cell_length, density);
    //
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(
        _buffer_float3.addr(), _num_constraints.addr(), _pos_t.addr()));

    auto end = thrust::make_zip_iterator(
        thrust::make_tuple(_buffer_float3.addr() + num,
                           _num_constraints.addr() + num, _pos_t.addr() + num));

    thrust::transform(
        thrust::device, begin, end, _pos_t.addr(),
        // particles->get_pos_ptr(), // Output to the positions buffer
        change_position_functor());

    iter++;
  }
}

__global__ void merge_mark_gpu(const int num, float3 *pos_granular,
                               float *mass_granular, float3 *vel_granular,
                               int *surface, float *surface_distance,
                               int *num_surface_neighbors, int *remove,
                               float *merge, float3 *merge_velocity,
                               int *cell_start_granular, float max_mass,
                               const int3 cell_size, const float cell_length) {
  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  if (i >= num)
    return;

  // Only non-surface particles that aren't already marked
  // if (surface[i] != 0 || atomicOr(&remove[i], 0) != 0 ||
  //     num_surface_neighbors[i] > 3)
  //   return;

  float mass_i = mass_granular[i];
  if (surface[i] != 0 || atomicOr(&remove[i], 0) != 0 || mass_i < 1.0f ||
      mass_i >= max_mass || surface_distance[i] < 100.0f) {
    return;
  }

  float3 pos_i = pos_granular[i];
  float3 vel_i = vel_granular[i];
  float closest_dis = 1000.0f;
  int closest_index = -1;

#pragma unroll
  for (auto m = 0; m < 27; __syncthreads(), ++m) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos_i / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);

    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;

    int j = cell_start_granular[cellID];
    while (j < cell_start_granular[cellID + 1] && j < num) {
      // if (j <= i || atomicOr(&remove[j], 0) != 0 || surface[j] != 0 ||
      //     num_surface_neighbors[j] > 3) {
      //
      if (j <= i || atomicOr(&remove[j], 0) != 0 || surface[j] != 0 ||
          surface_distance[i] < 100.0f) {
        j++;
        continue;
      }

      // Only merge if combined mass is valid
      float combined_mass = mass_i + mass_granular[j];
      if (combined_mass > max_mass) {
        j++;
        continue;
      }

      float3 p_j = pos_granular[j];
      float dis = length(pos_i - p_j);
      // float dis = norm3df(pos_i.x - p_j.x, pos_i.y - p_j.y, pos_i.z - p_j.z);

      if (dis < 0.035) {
        if (dis < closest_dis) {
          closest_dis = dis;
          closest_index = j;
        }
      }
      j++;
    }
  }

  // Only proceed if we found a merge candidate
  if (closest_index != -1) {
    if (atomicCAS(&remove[i], 0, -1) == 0 && surface[i] == 0 &&
        surface_distance[i] > 100.0f) {
      if (atomicCAS(&remove[closest_index], 0, -1) == 0 &&
          surface[closest_index] == 0 &&
          surface_distance[closest_index] > 100.0f) {
        atomicExch(&merge[closest_index], mass_i);
        atomicExch(&merge[i], -mass_i);

        const float m_t = 1 / (mass_i + mass_granular[closest_index]);
        const float3 vel_t =
            m_t * (mass_i * vel_i + mass_granular[closest_index] +
                   vel_granular[closest_index]);

        atomicExch(&merge_velocity[closest_index].x, vel_t.x);
        atomicExch(&merge_velocity[closest_index].y, vel_t.y);
        atomicExch(&merge_velocity[closest_index].z, vel_t.z);

        atomicExch(&merge_velocity[i].x, -vel_t.x);
        atomicExch(&merge_velocity[i].y, -vel_t.y);
        atomicExch(&merge_velocity[i].z, -vel_t.z);

        // printf("Merge set up: particle %d (mass %.3f) -> particle %d (mass "
        //        "%.3f)\n",
        //        i, mass_i, closest_index, mass_granular[closest_index]);
      } else {
        atomicExch(&remove[i], 0);
      }
    }
  }
}
__device__ bool isAlmostZero(float x) {
  return fabsf(x) < EPSILON_m; // fabsf for single precision
}

__global__ void merge_count_gpu(const int num, float *mass_del,
                                float *mass_granular, float3 *vel_granular,
                                int *merge_count, int *remove, float3 *vel_del,
                                const int blend_factor,
                                int *adaptive_last_step) {
  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  if (i >= num)
    return;

  if (remove[i] == -1) {
    int old_count = atomicAdd(&merge_count[i], 1);
    const float delta = mass_del[i] / blend_factor;
    const float3 delta_vel = vel_del[i] / blend_factor;
    mass_granular[i] += delta;
    adaptive_last_step[i] = 1;

    // vel_granular[i] += delta_vel;

    // Debug
    // printf("Particle %d: step %d/%d, mass %.3f, delta %.3f\n", i, old_count +
    // 1,
    //        blend_factor, mass_granular[i], delta);

    if (old_count + 1 == blend_factor) {
      if (mass_del[i] < 0) {
        // Only mark for removal if this particle is giving mass
        atomicExch(&remove[i], 1);
        adaptive_last_step[i] = 0;
        // printf("Particle %d marked for removal (final mass %.3f)\n", i,
        //        mass_granular[i]);
      } else {
        atomicExch(&remove[i], 0);
        // printf("Particle %d merge complete (final mass %.3f)\n", i,
        // mass_granular[i]);
      }
      atomicExch(&merge_count[i], 0);
      atomicExch(&mass_del[i], 0.0f);
      // vel_granular[i].x = 0;
      // vel_granular[i].y = 0;
      // vel_granular[i].z = 0;
    }
  }
}

struct SplitParticle {
  float3 pos;
  float3 vel;
  float mass;
  bool valid;
};

__device__ bool
check_neighborhood(float3 pos, float3 *pos_granular, float3 *pos_boundary,
                   int *cell_start_granular, int *cell_start_boundary,
                   const int3 cell_size, const float cell_length,
                   const float r_i, float3 &empty_cell_center,
                   int &neighbor_count) {
  neighbor_count = 0;
  bool found_empty = false;
  const float max_dist = 2.0f * r_i;

  // Loop through the 27 neighboring cells
  for (int m = 0; m < 27; m++) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);

    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;

    int j = cell_start_granular[cellID];

    // Check if cell is empty (but not the center cell)
    if (!found_empty && m != 13) { // m == 13 is the center cell
      if (cell_start_granular[cellID] == cell_start_granular[cellID + 1] &&
          cell_start_boundary[cellID] == cell_start_boundary[cellID + 1]) {
        // Calculate cell center position
        int3 cell_pos = make_int3(pos / cell_length) +
                        make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1);
        empty_cell_center = make_float3((cell_pos.x + 0.5f) * (cell_length),
                                        (cell_pos.y + 0.5f) * (cell_length),
                                        (cell_pos.z + 0.5f) * (cell_length));

        // const float dis = length(pos - empty_cell_center);

        found_empty = true;
      }
    }

    // Count granular neighbors in this cell
    // int j = cell_start_granular[cellID];
    // while (j < cell_start_granular[cellID + 1]) {
    //   const float3 pos_j = pos_granular[j];
    //   const float dis = length(pos - pos_j);
    //   if (dis > 0.0f && dis < max_dist) { // Exclude self
    //     neighbor_count++;
    //     if (neighbor_count >= 5) {
    //       return false; // Too many neighbors
    //     }
    //   }
    //   j++;
    // }

    // // Count boundary neighbors in this cell
    // j = cell_start_boundary[cellID];
    // while (j < cell_start_boundary[cellID + 1]) {
    //   const float3 pos_j = pos_boundary[j];
    //   const float dis = length(pos - pos_j);
    //   if (dis < max_dist) {
    //     neighbor_count++;
    //     if (neighbor_count >= 2) {
    //       return false; // Too many neighbors
    //     }
    //   }
    //   j++;
    // }
  }

  return found_empty; // Must have found an empty cell and have fewer than 5
                      // neighbors
}

__global__ void split_gpu(const int num, float3 *pos_granular,
                          float *mass_granular, float3 *vel_granular,
                          float3 *pos_boundary, int *surface, int *remove,
                          float *merge, int *cell_start_granular,
                          int *cell_start_boundary, const float max_mass,
                          const int3 cell_size, SplitParticle *split_particles,
                          int *split_count, const float density,
                          const float cell_length, int *adaptive_last_step) {

  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  if (i >= num)
    return;

  // Check if particle is marked for removal or is a surface particle
  if (atomicOr(&remove[i], 0) != 0 || surface[i] != 1 ||
      mass_granular[i] <= 2.0f) {
    return;
  }

  const float r_i = cbrtf((3 * mass_granular[i]) / (4 * PI * density));

  // Check neighborhood in a single pass
  float3 empty_cell_center;
  int neighbor_count;
  if (!check_neighborhood(pos_granular[i], pos_granular, pos_boundary,
                          cell_start_granular, cell_start_boundary, cell_size,
                          cell_length, r_i, empty_cell_center,
                          neighbor_count)) {
    return; // Either too many neighbors or no empty cells
  }

  // Atomic operation to reserve space for new particle
  int new_idx = atomicAdd(split_count, 1);

  // Create new particle
  SplitParticle new_particle;
  new_particle.mass = mass_granular[i] / 2.0f;
  new_particle.vel = vel_granular[i];
  new_particle.pos = empty_cell_center;
  new_particle.valid = true;

  // Update original particle
  mass_granular[i] = new_particle.mass;

  // Store new particle data
  split_particles[new_idx] = new_particle;
  adaptive_last_step[i] = 1;
}
// Define the extraction kernel properly
__global__ void extract_split_particles_kernel(SplitParticle *splits,
                                               float *masses, float3 *positions,
                                               float3 *velocities,
                                               int split_count) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= split_count)
    return;

  if (splits[idx].valid) {
    masses[idx] = splits[idx].mass;
    positions[idx] = splits[idx].pos;
    velocities[idx] = splits[idx].vel;
  }
}

struct merge_functor {

  merge_functor() {}

  __host__ __device__ float3
  operator()(const thrust::tuple<float, float> &t) const {
    const float &mass = thrust::get<0>(t);
  }
};

// Host function
void Solver::adaptive_sampling(
    std::shared_ptr<GranularParticles> &particles,
    const std::shared_ptr<GranularParticles> &boundaries,
    const DArray<int> &cell_start_granular,
    const DArray<int> &cell_start_boundary, const float max_mass,
    int3 cell_size, float3 space_size, float cell_length, const float density) {
  const int num = particles->size();
  if (num == 0)
    return;

  // Store initial state and particle IDs
  std::vector<float> initial_masses(num);
  std::vector<float3> initial_positions(num);
  std::vector<float3> initial_velocities(num);
  std::vector<int> remove_flags(num);
  std::vector<float> merge_values(num);

  // Copy initial data to host
  CUDA_CALL(hipMemcpy(initial_masses.data(), particles->get_mass_ptr(),
                       sizeof(float) * num, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(initial_positions.data(), particles->get_pos_ptr(),
                       sizeof(float3) * num, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(initial_velocities.data(), particles->get_vel_ptr(),
                       sizeof(float3) * num, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(remove_flags.data(), _buffer_remove.addr(),
                       sizeof(int) * num, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(merge_values.data(), _buffer_merge.addr(),
                       sizeof(float) * num, hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  try {
    // old mass

    auto m_t = thrust::device_pointer_cast(particles->get_mass_ptr());

    DArray<float> old_masses(particles->size());
    CUDA_CALL(hipMemcpy(old_masses.addr(), particles->get_mass_ptr(),
                         sizeof(float) * particles->size(),
                         hipMemcpyDeviceToDevice));

    const float old_mass =
        thrust::reduce(m_t, m_t + num, 0, thrust::plus<float>());

    // Run merge kernel
    if (t_merge_iter == 5) {
      merge_mark_gpu<<<(num + block_size - 1) / block_size, block_size>>>(
          num, particles->get_pos_ptr(), particles->get_mass_ptr(),
          particles->get_vel_ptr(), particles->get_surface_ptr(),
          particles->get_surface_distance_ptr(),
          particles->get_num_surface_ptr(), _buffer_remove.addr(),
          _buffer_merge.addr(), _buffer_merge_velocity.addr(),
          cell_start_granular.addr(), max_mass, cell_size, cell_length);

      t_merge_iter = 0;
    }
    t_merge_iter++;

    // Print info about particles being removed
    // std::vector<int> host_remove(num);
    // std::vector<float> host_mass(num);
    // CUDA_CALL(hipMemcpy(host_remove.data(), _buffer_remove.addr(),
    //                      sizeof(int) * num, hipMemcpyDeviceToHost));
    // CUDA_CALL(hipMemcpy(host_mass.data(), particles->get_mass_ptr(),
    //                      sizeof(float) * num, hipMemcpyDeviceToHost));

    // int removal_count = 0;
    // std::cout << "Particles marked for removal (mass):\n";
    // for (int i = 0; i < num; i++) {
    //   if (host_remove[i] == 1) {
    //     removal_count++;
    //     std::cout << "Particle " << i << ": mass = " << host_mass[i]
    //               << ", remove flag = " << host_remove[i]
    //               << ", merge mass delta = ";

    //     // Get merge mass delta for this particle
    //     float merge_delta;
    //     CUDA_CALL(hipMemcpy(&merge_delta, &_buffer_merge.addr()[i],
    //                          sizeof(float), hipMemcpyDeviceToHost));
    //     std::cout << merge_delta << "\n";
    //   }
    // }
    // std::cout << "Total particles marked for removal: " << removal_count
    //           << "\n";

    // Run split kernel

    // TODO: Velocity update
    // thrust::transform(thrust::device, particles->get_mass_ptr(),
    //                   particles->get_mass_ptr() + num,
    //                   _buffer_merge.addr(), particles->get_mass_ptr(),
    //                   thrust::plus<float>());
    //

    CUDA_CALL(hipDeviceSynchronize());

    merge_count_gpu<<<(num + block_size - 1) / block_size, block_size>>>(
        num, _buffer_merge.addr(), particles->get_mass_ptr(),
        particles->get_vel_ptr(), _buffer_merge_count.addr(),
        _buffer_remove.addr(), _buffer_merge_velocity.addr(), _blend_factor,
        particles->get_adaptive_last_step_ptr());

    CUDA_CALL(hipDeviceSynchronize());

    // Create and initialize split counter on device
    // Allocate space for split particles
    DArray<SplitParticle> split_particles(num); // Maximum possible splits
    int host_split_count = 0;

    int *d_split_count;
    CUDA_CALL(hipMalloc(&d_split_count, sizeof(int)));
    CUDA_CALL(hipMemcpy(d_split_count, &host_split_count, sizeof(int),
                         hipMemcpyHostToDevice));
    // TODO: fix the split kernel
    // Run split kernel
    split_gpu<<<(num + block_size - 1) / block_size, block_size>>>(
        num, particles->get_pos_ptr(), particles->get_mass_ptr(),
        particles->get_vel_ptr(), boundaries->get_pos_ptr(),
        particles->get_surface_ptr(), _buffer_remove.addr(),
        _buffer_merge.addr(), cell_start_granular.addr(),
        cell_start_boundary.addr(), max_mass, cell_size, split_particles.addr(),
        d_split_count, density, cell_length,
        particles->get_adaptive_last_step_ptr());

    // // Print final state before removal
    // std::cout << "\nFinal state before removal:\n";
    // CUDA_CALL(hipMemcpy(host_mass.data(), particles->get_mass_ptr(),
    //                      sizeof(float) * num, hipMemcpyDeviceToHost));
    // CUDA_CALL(hipMemcpy(host_remove.data(), _buffer_remove.addr(),
    //                      sizeof(int) * num, hipMemcpyDeviceToHost));
    //
    // for (int i = 0; i < num; i++) {
    //   if (host_remove[i] == 1) {
    //     std::cout << "Particle " << i << ": final mass = " << host_mass[i]
    //               << "\n";
    //   }
    // }
    //
    // remove elements
    // Copy arrays from device to host for checking mass changes
    // std::vector<int> host_remove(particles->size());
    // std::vector<float> host_old_masses(particles->size());
    // std::vector<int> host_merge(particles->size());
    // std::vector<float> host_current_masses(particles->size());

    // CUDA_CALL(hipMemcpy(host_remove.data(), _buffer_remove.addr(),
    //                      sizeof(int) * particles->size(),
    //                      hipMemcpyDeviceToHost));
    // CUDA_CALL(hipMemcpy(host_old_masses.data(), old_masses.addr(),
    //                      sizeof(float) * particles->size(),
    //                      hipMemcpyDeviceToHost));
    // CUDA_CALL(hipMemcpy(host_merge.data(), _buffer_merge.addr(),
    //                      sizeof(float) * particles->size(),
    //                      hipMemcpyDeviceToHost));
    // CUDA_CALL(hipMemcpy(host_current_masses.data(),
    // particles->get_mass_ptr(),
    //                      sizeof(float) * particles->size(),
    //                      hipMemcpyDeviceToHost));

    // for (int i = 0; i < particles->size(); i++) {
    //   if (host_remove[i] == 0) {
    //     // Print masses that changed when they shouldn't have
    //     if (host_old_masses[i] != host_current_masses[i]) {
    //       std::cout << "Particle " << i << " mass changed unexpectedly from "
    //                 << host_old_masses[i] << " to " << host_current_masses[i]
    //                 << std::endl;
    //     }
    //   }
    // }

    CUDA_CALL(hipDeviceSynchronize());

    try {
      particles->remove_elements(_buffer_remove);
    } catch (const std::exception &e) {
      std::cerr << "Error compacting particles: " << e.what() << std::endl;
      throw;
    }

    try {
      _buffer_merge_count.compact(_buffer_remove);
    } catch (const std::exception &e) {
      std::cerr << "Error compacting merge count: " << e.what() << std::endl;
      throw;
    }

    try {
      _buffer_merge.compact(_buffer_remove);
    } catch (const std::exception &e) {
      std::cerr << "Error compacting merge buffer: " << e.what() << std::endl;
      throw;
    }

    try {
      _buffer_remove.compact(_buffer_remove);
    } catch (const std::exception &e) {
      std::cerr << "Error compacting remove buffer: " << e.what() << std::endl;
      throw;
    }

    CUDA_CALL(hipDeviceSynchronize());
    // Get new size after compacting
    const int new_num = particles->size();

    // Get final state
    std::vector<float> final_masses(new_num);
    std::vector<float3> final_positions(new_num);
    std::vector<float3> final_velocities(new_num);

    CUDA_CALL(hipMemcpy(final_masses.data(), particles->get_mass_ptr(),
                         sizeof(float) * new_num, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(final_positions.data(), particles->get_pos_ptr(),
                         sizeof(float3) * new_num, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(final_velocities.data(), particles->get_vel_ptr(),
                         sizeof(float3) * new_num, hipMemcpyDeviceToHost));

    // // First, create a mapping of original to new indices
    // std::vector<int> index_mapping(num, -1); // Initialize all to -1
    // int new_idx = 0;
    // for (int i = 0; i < num; i++) {
    //   if (remove_flags[i] != 1) { // If particle survives
    //     index_mapping[i] = new_idx++;
    //   }
    // }

    // // Now verify ordering is maintained
    // bool ordering_maintained = true;
    // for (int old_idx = 0; old_idx < num; old_idx++) {
    //   if (remove_flags[old_idx] != 1) { // If this particle survived
    //     int new_idx = index_mapping[old_idx];

    //     // Check mass, accounting for merge values
    //     if (std::abs(final_masses[new_idx] - initial_masses[old_idx]) > 1e-6
    //     &&
    //         std::abs(final_masses[new_idx] - (initial_masses[old_idx] +
    //                                           merge_values[old_idx])) > 1e-6)
    //                                           {
    //       std::cout << "Mass mismatch for particle that moved from " <<
    //       old_idx
    //                 << " to " << new_idx << std::endl;
    //       std::cout << "Original mass: " << initial_masses[old_idx]
    //                 << std::endl;
    //       std::cout << "Merge value: " << merge_values[old_idx] << std::endl;
    //       std::cout << "Final mass: " << final_masses[new_idx] << std::endl;
    //       // ordering_maintained = false;
    //     }

    //     // Check position
    //     if (length(final_positions[new_idx] - initial_positions[old_idx]) >
    //         1e-6) {
    //       std::cout << "Position mismatch for particle that moved from "
    //                 << old_idx << " to " << new_idx << std::endl;
    //       std::cout << "Original position: (" << initial_positions[old_idx].x
    //                 << ", " << initial_positions[old_idx].y << ", "
    //                 << initial_positions[old_idx].z << ")" << std::endl;
    //       std::cout << "Final position: (" << final_positions[new_idx].x <<
    //       ", "
    //                 << final_positions[new_idx].y << ", "
    //                 << final_positions[new_idx].z << ")" << std::endl;
    //       ordering_maintained = false;
    //     }

    //     // Check velocity
    //     if (length(final_velocities[new_idx] - initial_velocities[old_idx]) >
    //         1e-6) {
    //       std::cout << "Velocity mismatch for particle that moved from "
    //                 << old_idx << " to " << new_idx << std::endl;
    //       std::cout << "Original velocity: (" <<
    //       initial_velocities[old_idx].x
    //                 << ", " << initial_velocities[old_idx].y << ", "
    //                 << initial_velocities[old_idx].z << ")" << std::endl;
    //       std::cout << "Final velocity: (" << final_velocities[new_idx].x
    //                 << ", " << final_velocities[new_idx].y << ", "
    //                 << final_velocities[new_idx].z << ")" << std::endl;
    //       ordering_maintained = false;
    //     }
    //   }
    // }

    // if (!ordering_maintained) {
    //   std::cout << "WARNING: Particle property consistency not maintained "
    //                "after compacting!"
    //             << std::endl;
    // } else {
    //   std::cout << "Particle property consistency maintained successfully."
    //             << std::endl;
    // }
    // add elements

    // Get number of splits
    // TODO : check order of split
    CUDA_CALL(hipMemcpy(&host_split_count, d_split_count, sizeof(int),
                         hipMemcpyDeviceToHost));
    CUDA_CALL(hipFree(d_split_count));

    if (host_split_count > 0) {
      // Prepare arrays for new particles
      DArray<float> new_masses(host_split_count);
      DArray<float3> new_positions(host_split_count);
      DArray<float3> new_velocities(host_split_count);

      // Create zero-filled arrays for buffers
      DArray<int> new_remove(host_split_count);
      DArray<int> new_merge_count(host_split_count);
      DArray<float> new_merge(host_split_count);

      // Fill new arrays with zeros
      thrust::fill(
          thrust::device, thrust::device_pointer_cast(new_remove.addr()),
          thrust::device_pointer_cast(new_remove.addr() + host_split_count), 0);
      thrust::fill(thrust::device,
                   thrust::device_pointer_cast(new_merge_count.addr()),
                   thrust::device_pointer_cast(new_merge_count.addr() +
                                               host_split_count),
                   0);
      thrust::fill(
          thrust::device, thrust::device_pointer_cast(new_merge.addr()),
          thrust::device_pointer_cast(new_merge.addr() + host_split_count),
          0.0f);

      // Extract split particles

      dim3 block(256);
      dim3 grid((host_split_count + block.x - 1) / block.x);

      extract_split_particles_kernel<<<grid, block>>>(
          split_particles.addr(), new_masses.addr(), new_positions.addr(),
          new_velocities.addr(), host_split_count);

      CUDA_CALL(hipDeviceSynchronize());
      CHECK_KERNEL();

      // Add new particles
      particles->add_elements(new_masses, new_positions, new_velocities,
                              host_split_count);

      // Append zeros to buffers
      _buffer_remove.append(new_remove);
      _buffer_merge_count.append(new_merge_count);
      _buffer_merge.append(new_merge);

      // Verify sizes match
      if (_buffer_remove.length() != particles->size() ||
          _buffer_merge_count.length() != particles->size() ||
          _buffer_merge.length() != particles->size()) {
        throw std::runtime_error(
            "Buffer sizes don't match particle count after split");
      }
    }
    // change in mass

    // Print total mass after
    // auto m_t_n = thrust::device_pointer_cast(particles->get_mass_ptr());
    // const float new_mass = thrust::reduce(m_t_n, m_t_n +
    // particles->size(), 0,
    //                                       thrust::plus<float>());
    // std::cout << "Total mass after: " << new_mass << "\n";
    // if ((new_mass - old_mass) != 0) {
    //   std::cout << "Change in mass " << new_mass - old_mass << "\n";
    // }
    //

  } catch (const std::exception &e) {
    std::cerr << "Error in adaptive_sampling: " << e.what() << std::endl;
    return;
  }
}

__global__ void update_upsampled_cuda(
    float3 *pos_upsampled, float3 *pos_granular, float3 *pos_boundary,
    float3 *vel_granular, float3 *vel_upsampled, const int n,
    int *cell_start_upsampled, int *cell_start_granular,
    int *cell_start_boundary, const int3 cell_size, const float cell_length) {

  const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  if (i >= n)
    return;

  const float3 pos_i = pos_upsampled[i];
  float3 weighted_vel = make_float3(0.0f, 0.0f, 0.0f);
  float granular_weight = 0.0f;
  float boundary_weight = 0.0f;
  float max_w_ij = 0.0f;
  const float3 g_t = make_float3(0.0f, -9.8f, 0.0f);
  const float d_t = 0.002f;

  // Boundary repulsion parameters
  const float boundary_radius = 0.02f;
  const float repulsion_strength = 5.0f;
  float3 boundary_repulsion = make_float3(0.0f, 0.0f, 0.0f);

#pragma unroll
  for (auto m = 0; m < 27; __syncthreads(), ++m) {
    const auto cellID = particlePos2cellIdx(
        make_int3(pos_i / cell_length) +
            make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
        cell_size);

    if (cellID == (cell_size.x * cell_size.y * cell_size.z))
      continue;

    // Handle granular particles
    int j = cell_start_granular[cellID];
    while (j < cell_start_granular[cellID + 1]) {
      const float dis = length(pos_i - pos_granular[j]);
      const float t_1 = 1 - (dis * dis * r_9);

      const float w_ij = max(0.0f, t_1 * t_1 * t_1);
      granular_weight += w_ij;
      weighted_vel += w_ij * vel_granular[j];
      max_w_ij = max(max_w_ij, w_ij);
      j++;
    }

    // Handle boundary particles
    int k = cell_start_boundary[cellID];
    while (k < cell_start_boundary[cellID + 1]) {
      const float3 to_boundary = pos_i - pos_boundary[k];
      const float dis = length(to_boundary);

      // Add repulsion force when too close to boundary
      if (dis < boundary_radius) {
        float3 normal = to_boundary / (dis + 1e-6f);
        float force =
            repulsion_strength * (boundary_radius - dis) / boundary_radius;
        boundary_repulsion += normal * force;
      }

      // Calculate boundary influence
      const float t_1 = 1 - (dis * dis * r_9);
      const float w_ij = max(0.0f, t_1 * t_1 * t_1);
      boundary_weight += w_ij;

      k++;
    }
  }

  // Update velocity and position
  float3 new_vel;
  if (granular_weight > 0.0f) {
    // If there are nearby granular particles, use their influence
    weighted_vel /= granular_weight;
    float alpha = max(0.0f, 1.0f - max_w_ij);
    new_vel =
        alpha * (vel_upsampled[i] + g_t * d_t) + (1.0f - alpha) * weighted_vel;
  } else {
    // If no granular particles nearby, use current velocity with boundary
    // repulsion
    new_vel = vel_upsampled[i] * 0.9f + g_t * d_t;
  }

  // Add boundary repulsion to velocity
  new_vel += boundary_repulsion;

  // Update velocity and position
  vel_upsampled[i] = new_vel;
  pos_upsampled[i] = pos_upsampled[i] + new_vel * d_t;

  // Boundary constraints
  if (pos_upsampled[i].y < 0.005f) {
    pos_upsampled[i].y = 0.005f;
    vel_upsampled[i].y = max(0.0f, vel_upsampled[i].y);
  }

  // if (pos_upsampled[i].x > 1.95) {
  //   pos_upsampled[i].x = 1.95;
  // }
  // if (pos_upsampled[i].x < 0.05) {
  //   pos_upsampled[i].x = 0.05;
  // }
  // if (pos_upsampled[i].z > 1.75) {
  //   pos_upsampled[i].z = 1.75;
  // }
  // if (pos_upsampled[i].z < 0.05) {
  //   pos_upsampled[i].z = 0.05;
  // }

  return;
}

void Solver::upsampled_update(
    std::shared_ptr<GranularParticles> &particles,
    const std::shared_ptr<GranularParticles> &boundaries,
    std::shared_ptr<GranularParticles> &upsampled,
    const DArray<int> &cell_start_upsampled,
    const DArray<int> &cell_start_granular,
    const DArray<int> &cell_start_boundary, int3 cell_size, float3 space_size,
    float cell_length, const float density) {
  const int num = upsampled->size();

  update_upsampled_cuda<<<(num + block_size - 1) / block_size, block_size>>>(
      upsampled->get_pos_ptr(), particles->get_pos_ptr(),
      boundaries->get_pos_ptr(), particles->get_vel_ptr(),
      upsampled->get_vel_ptr(), num, cell_start_upsampled.addr(),
      cell_start_granular.addr(), cell_start_boundary.addr(), cell_size,
      cell_length);
}

__global__ void apply_explosion_force(float3 *pos, float3 *vel, float *mass,
                                      float3 center_of_mass,
                                      float explosion_force,
                                      int num_particles) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_particles)
    return;

  // Calculate direction from center of mass to particle
  float3 direction = pos[idx] - center_of_mass;
  float distance = length(direction);

  if (distance < EPSILON_m)
    return; // Avoid division by zero

  // Normalize direction
  direction = direction / distance;

  // Force decreases with square of distance
  float force_magnitude = explosion_force / (1.0f + distance * distance);

  // Apply force as velocity change
  float3 velocity_change = direction * force_magnitude / mass[idx];
  vel[idx] += velocity_change;
}

void Solver::trigger_explosion(std::shared_ptr<GranularParticles> &particles,
                               float explosion_force) {
  int num = particles->size();
  if (num == 0)
    return;

  // Calculate center of mass
  float3 center_of_mass = make_float3(0.0f, 0.0f, 0.0f);
  float total_mass = 0.0f;

  // Use thrust to calculate center of mass
  thrust::device_ptr<float3> pos_ptr(particles->get_pos_ptr());
  thrust::device_ptr<float> mass_ptr(particles->get_mass_ptr());

  for (int i = 0; i < num; i++) {
    float mass = mass_ptr[i];
    float3 pos = pos_ptr[i];
    center_of_mass += make_float3(pos.x * mass, pos.y * mass, pos.z * mass);
    total_mass += mass;
  }

  if (total_mass > 0) {
    center_of_mass = center_of_mass / total_mass;
  }

  // Apply explosion force
  apply_explosion_force<<<(num + block_size - 1) / block_size, block_size>>>(
      particles->get_pos_ptr(), particles->get_vel_ptr(),
      particles->get_mass_ptr(), center_of_mass, explosion_force, num);

  hipDeviceSynchronize();
}
